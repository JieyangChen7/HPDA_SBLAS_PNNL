#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "hipsparse.h"
#include <vector>
#include <iostream>
#include <cstdio>
#include <pthread.h>
#include "spmv_task.h"
#include "spmv_kernel.h"
#include <omp.h>
#include <hip/hip_runtime_api.h>
//#include "anonymouslib_cuda.h"

using namespace std;

void * spmv_worker(void * arg);

void generate_tasks(int m, int n, long long nnz, double * alpha,
				    double * csrVal, long long * csrRowPtr, int * csrColIndex, 
				  	double * x, double * beta,
				  	double * y,
				  	long long nb,
				  	vector<spmv_task *> * spmv_task_pool_ptr);

void assign_task(spmv_task * t, int dev_id, hipStream_t stream);

int run_task(spmv_task * t, int dev_id, hipsparseHandle_t handle, int kernel, hipStream_t stream);

void finalize_task(spmv_task * t, int dev_id, hipStream_t stream);

void gather_results(vector<spmv_task *> * spmv_task_completed, double * y, double * beta, int m);

void print_task_info(spmv_task * t);


spmv_ret spMspV_mgpu_v2_num(int m, int n, long long nnz, double * alpha,
                                  double * csrVal, long long * csrRowPtr, int * csrColIndex,
                                  double * x, double * beta,
                                  double * y,
                                  int ngpu,
                                  int kernel,
                                  long long nb,
                                  int q) {
  //cout << "start1\n";
  long long nnz_reduced = 0;
  vector<double> * csrVal_reduced = new vector<double>();
  vector<long long> * csrRowPtr_reduced = new vector<long long>();
  vector<int> * csrColIndex_reduced = new vector<int>();

  // cout << "start2\n";
  double * csrVal_reduced_pin;
  long long * csrRowPtr_reduced_pin;
  int * csrColIndex_reduced_pin;
 
  // cout << "start3\n";
  csrRowPtr_reduced->push_back(0);
  for (int i = 0; i < m; i++) {
    for (int j = csrRowPtr[i]; j < csrRowPtr[i+1]; j++) {
      if (x[csrColIndex[j]] != 0.0) {
        csrVal_reduced->push_back(csrVal[j]);
        csrColIndex_reduced->push_back(csrColIndex[j]);
        nnz_reduced ++;
      }
    }
    csrRowPtr_reduced->push_back(nnz_reduced);
  }


  // cout << "start4\n";
  hipHostMalloc((void **)&csrVal_reduced_pin, nnz_reduced * sizeof(double));
  hipHostMalloc((void **)&csrRowPtr_reduced_pin, (m+1) * sizeof(long long));
  hipHostMalloc((void **)&csrColIndex_reduced_pin, nnz_reduced * sizeof(int));
  
  // cout << "start5\n";
  hipMemcpy(csrVal_reduced_pin, csrVal_reduced->data(), nnz_reduced * sizeof(double), hipMemcpyHostToHost);
  hipMemcpy(csrRowPtr_reduced_pin, csrRowPtr_reduced->data(), (m+1) * sizeof(long long), hipMemcpyHostToHost);
  hipMemcpy(csrColIndex_reduced_pin, csrColIndex_reduced->data(), nnz_reduced * sizeof(int), hipMemcpyHostToHost);

  // cout << "start6\n";
  delete csrVal_reduced;
  delete csrRowPtr_reduced;
  delete csrColIndex_reduced;

  cout << "spMspV_mgpu_v2: nnz reduced from " << nnz << " to " << nnz_reduced
<< std::endl;

  spmv_ret ret = spMV_mgpu_v2(m, n, nnz_reduced, alpha,
               csrVal_reduced_pin,
               csrRowPtr_reduced_pin, 
               csrColIndex_reduced_pin,
               x, beta, y, ngpu, kernel,
	       nb, q);

  hipHostFree(csrVal_reduced_pin);
  hipHostFree(csrRowPtr_reduced_pin);
  hipHostFree(csrColIndex_reduced_pin);


  return ret;


}

spmv_ret spMV_mgpu_v2_numa(int m, int n, long long nnz, double * alpha,
				  double * csrVal, long long * csrRowPtr, int * csrColIndex, 
				  double * x, double * beta,
				  double * y,
				  int ngpu, 
				  int kernel,
				  long long nb,
				  int q,
					int * numa_mapping)
{

	
	nb = min(nb, (long long )(0.8*16*1e9/(double)(sizeof(double) + sizeof(int) + sizeof(int)))/q ); 
	//if (nb <= 0 || ngpu == 0 || q == 0) {
//		cout << "nb = " << nb << endl;
//		cout << "ngpu = " << ngpu << endl;
//		cout << "q = " << q << endl; 
		//return -1;
	//}


	double curr_time = 0.0;
	double time_parse = 0.0;
	double time_comm_comp = 0.0;
	double time_post = 0.0;

	curr_time = get_time();

	// figure out the number of numa nodes
	int num_numa_nodes = 0;
	for (int i = 0; i < ngpu; i++) {
		if (numa_mapping[i] > num_numa_nodes) {
			num_numa_nodes = numa_mapping[i];
		}
	}
	num_numa_nodes += 1; 
	int * representive_threads = new int [num_numa_nodes];
	for (int i = 0; i < num_numa_nodes; i++) {
		for (int j = 0; j < ngpu; j++) {
			if (numa_mapping[j] == i) {
				representive_threads[i] = j;
				break;
			}
		}
	}

	vector<spmv_task *> ** spmv_task_pool = new new vector<spmv_task *>*[num_numa_nodes];
	vector<spmv_task *> ** spmv_task_completed = new new vector<spmv_task *>*[num_numa_nodes];

	for (int i = 0; i < num_numa_nodes; i++) {
		spmv_task_pool[i] = new vector<spmv_task *>();
		spmv_task_completed[i] = new vector<spmv_task *>();
	}

	//vector<spmv_task *> * spmv_task_pool = new vector<spmv_task *>();
	//vector<spmv_task *> * spmv_task_completed = new vector<spmv_task *>();

	//generate_tasks(m, n, nnz, alpha,
	//			  csrVal, csrRowPtr, csrColIndex, 
	//			  x, beta, y, nb,
	//			  spmv_task_pool);

	//int num_of_tasks = (*spmv_task_pool).size();

	//(*spmv_task_completed).reserve(num_of_tasks);

	//time_parse = get_time() - curr_time;

	//curr_time = get_time();

	//hipProfilerStart();
	//cout << "starting " << ngpu << " GPUs." << endl;
	omp_set_num_threads(ngpu);
	//cout << "omp_get_max_threads = " << omp_get_max_threads() << endl;
	//cout << "omp_get_thread_limit = " << omp_get_thread_limit() << endl;
	double core_time;
	#pragma omp parallel default (shared) reduction(max:core_time)
	{



		int c;
		unsigned int dev_id = omp_get_thread_num();

		// generate tasks on each NUMA node
    for (int i = 0; i < num_numa_nodes; i++) {
      if (representive_threads[i] == dev) {
				long long  start_idx, end_idx;
					

			}

    }


		//cout << "thread " << dev_id <<"/" << omp_get_num_threads() << "started" << endl;
		hipSetDevice(dev_id);
		
		hipsparseStatus_t status[q];
		hipStream_t stream[q];
		hipsparseHandle_t handle[q];

		

		double ** dev_csrVal = new double * [q];
		int ** dev_csrRowPtr = new int    * [q];
		int ** dev_csrColIndex = new int  * [q];
		double ** dev_x = new double      * [q];
		double ** dev_y = new double      * [q];

		for (c = 0; c < q; c++) {
			hipStreamCreate(&(stream[c]));
			status[c] = hipsparseCreate(&(handle[c])); 
			if (status[c] != HIPSPARSE_STATUS_SUCCESS) 
			{ 
				printf("CUSPARSE Library initialization failed");
				//return 1; 
			} 
			status[c] = hipsparseSetStream(handle[c], stream[c]);
			if (status[c] != HIPSPARSE_STATUS_SUCCESS) 
			{ 
				printf("Stream bindind failed");
				//return 1;
			} 

			hipMalloc((void**)&(dev_csrVal[c]),      nb      * sizeof(double));
			hipMalloc((void**)&(dev_csrRowPtr[c]),   (m + 1) * sizeof(int)   );
			hipMalloc((void**)&(dev_csrColIndex[c]), nb      * sizeof(int)   );
			hipMalloc((void**)&(dev_x[c]),           n       * sizeof(double));
		  hipMalloc((void**)&(dev_y[c]),           m       * sizeof(double));

    }

   	c = 0; 
    	
    //cout << "GPU " << dev_id << " entering loop" << endl;


    int num_of_assigned_task = 0;
    int num_of_to_be_assigned_task = num_of_tasks * (dev_id + 1) /  omp_get_num_threads() - 
    									 num_of_tasks * (dev_id) /  omp_get_num_threads();
		//hipProfilerStart();
	
		// for debug
		#pragma omp barrier
		double profile_time = get_time();
		int iter = 0;
		while (true) {

			spmv_task * curr_spmv_task;

			for (c = 0; c < q; c++) {

				//printf("GPU[%d] c[%d] iter[%d] time[%f]: try to get one task\n", dev_id, c, iter, get_time() - profile_time);
				#pragma omp critical
				{
					//printf("GPU[%d] c[%d] iter[%d] time[%f]: enter critial region\n", dev_id, c, iter, get_time() - profile_time);
					if(num_of_assigned_task < num_of_to_be_assigned_task &&
						 (*spmv_task_pool).size() > 0) {
						curr_spmv_task = (*spmv_task_pool)[(*spmv_task_pool).size() - 1];
						(*spmv_task_pool).pop_back();
						(*spmv_task_completed).push_back(curr_spmv_task);
						num_of_assigned_task++;
						//printf("GPU[%d] c[%d] iter[%d] time[%f]: got one task assigned[%d] / to be assigned[%d] / total [%d] )\n", dev_id, c, iter, get_time() - profile_time, num_of_assigned_task, num_of_to_be_assigned_task, (*spmv_task_pool).size());
					} else {
						curr_spmv_task = NULL;
					}
				}

				//printf("GPU[%d] c[%d] iter[%d] time[%f]: got out of critical region\n", dev_id, c, iter, get_time() - profile_time);
				if (curr_spmv_task) {

					curr_spmv_task->dev_csrVal = dev_csrVal[c];
					curr_spmv_task->dev_csrRowPtr = dev_csrRowPtr[c];
					curr_spmv_task->dev_csrColIndex = dev_csrColIndex[c];
					curr_spmv_task->dev_x = dev_x[c];
					curr_spmv_task->dev_y = dev_y[c];
			
					//hipEventCreate(&start);
					//hipEventCreate(&stop);
	
					//printf("GPU[%d] c[%d] iter[%d] time[%f]: assign task\n", dev_id, c, iter, get_time() - profile_time);
					assign_task(curr_spmv_task, dev_id, stream[c]);
					//printf("GPU[%d] c[%d] iter[%d] time[%f]: run task\n", dev_id, c, iter, get_time() - profile_time);
					run_task(curr_spmv_task, dev_id, handle[c], kernel, stream[c]);
					//printf("GPU[%d] c[%d] iter[%d] time[%f]: finalize task\n", dev_id, c, iter, get_time() - profile_time);
					finalize_task(curr_spmv_task, dev_id, stream[c]);
					//printf("GPU[%d] c[%d] iter[%d] time[%f]: done processing a task\n", dev_id, c, iter, get_time() - profile_time);
				}
				if (!curr_spmv_task) {
					break;
				}

			}
			if (!curr_spmv_task) {
				break;
			}
			//printf("GPU[%d] c[%d] iter[%d] time[%f]: start sync\n", dev_id, c, iter, get_time() - profile_time);
			for (c = 0; c < q; c++) {
			    hipStreamSynchronize(stream[c]);
			}
			iter++;
			//printf("GPU[%d] c[%d] iter[%d] time[%f]: sync complete\n", dev_id, c, iter, get_time() - profile_time);
			if (num_of_to_be_assigned_task == 0) {
				break;
			}
		} //end of while
		core_time = get_time() - profile_time;
		//hipDeviceSynchronize();
		//hipProfilerStop();
		for (c = 0; c < q; c++) {

			hipFree(dev_csrVal[c]);
			hipFree(dev_csrRowPtr[c]);
			hipFree(dev_csrColIndex[c]);
			hipFree(dev_x[c]);
			hipFree(dev_y[c]);
			hipsparseDestroy(handle[c]);
			hipStreamDestroy(stream[c]);
		}

		


	} // end of omp

	time_comm_comp = core_time;

	curr_time = get_time();

	gather_results(spmv_task_completed, y, beta, m);

	for (int t = 0; t < (*spmv_task_completed).size(); t++) {
		hipHostFree((*spmv_task_completed)[t]->host_csrRowPtr);
		hipHostFree((*spmv_task_completed)[t]->local_result_y);
		hipHostFree((*spmv_task_completed)[t]->alpha);
		hipHostFree((*spmv_task_completed)[t]->beta);

	}

	time_post = get_time() - curr_time;
	spmv_ret ret;
        ret.comp_time = time_comm_comp;
	return ret;
	//cout << "time_parse = " << time_parse << ", time_comm_comp = " << time_comm_comp << ", time_post = " << time_post << endl;
}



void generate_tasks(int m, int n, long long nnz, double * alpha,
				    double * csrVal, long long * csrRowPtr, int * csrColIndex, 
				  	double * x, double * beta,
				  	double * y,
				  	long long nb,
				  	vector<spmv_task *> * spmv_task_pool_ptr) {

	int num_of_tasks = (int)((nnz + nb - 1) / nb);
	//cout << "num_of_tasks = " << num_of_tasks << endl;

	int curr_row;
	int t;
	int d;

	spmv_task * spmv_task_pool = new spmv_task[num_of_tasks];

	// Calculate the start and end index
	for (t = 0; t < num_of_tasks; t++) {
		long long tmp1 = t * nnz;
		long long tmp2 = (t + 1) * nnz;

		double tmp3 = (double)(tmp1 / num_of_tasks);
		double tmp4 = (double)(tmp2 / num_of_tasks);

		spmv_task_pool[t].start_idx = floor((double)(tmp1 / num_of_tasks));
		spmv_task_pool[t].end_idx   = floor((double)(tmp2 / num_of_tasks)) - 1;
		spmv_task_pool[t].dev_nnz = (int)(spmv_task_pool[t].end_idx - spmv_task_pool[t].start_idx + 1);
	}

	// Calculate the start and end row
	curr_row = 0;
	for (t = 0; t < num_of_tasks; t++) {

		spmv_task_pool[t].start_row = get_row_from_index(m, csrRowPtr, spmv_task_pool[t].start_idx);
		// Mark imcomplete rows
		// True: imcomplete
		if (spmv_task_pool[t].start_idx > csrRowPtr[spmv_task_pool[t].start_row]) {
			spmv_task_pool[t].start_flag = true;
			spmv_task_pool[t].y2 = y[spmv_task_pool[t].start_row];
		} else {
			spmv_task_pool[t].start_flag = false;
		}
	}

	curr_row = 0;
	for (t = 0; t < num_of_tasks; t++) {
		spmv_task_pool[t].end_row = get_row_from_index(m, csrRowPtr, spmv_task_pool[t].end_idx);

		// Mark imcomplete rows
		// True: imcomplete
		if (spmv_task_pool[t].end_idx < csrRowPtr[spmv_task_pool[t].end_row + 1] - 1)  {
			spmv_task_pool[t].end_flag = true;
			spmv_task_pool[t].y2 = y[spmv_task_pool[t].end_row];
		} else {
			spmv_task_pool[t].end_flag = false;
		}
	}

	// Cacluclate dimensions
	for (t = 0; t < num_of_tasks; t++) {
		spmv_task_pool[t].dev_m = spmv_task_pool[t].end_row - spmv_task_pool[t].start_row + 1;
		spmv_task_pool[t].dev_n = n;
	}


	for (t = 0; t < num_of_tasks; t++) {
		hipHostMalloc((void **)&(spmv_task_pool[t].host_csrRowPtr), (spmv_task_pool[t].dev_m + 1) * sizeof(int));

		spmv_task_pool[t].host_csrRowPtr[0] = 0;
		spmv_task_pool[t].host_csrRowPtr[spmv_task_pool[t].dev_m] = spmv_task_pool[t].dev_nnz;
	
		// memcpy(&(spmv_task_pool[t].host_csrRowPtr[1]), 
		// 	   &csrRowPtr[spmv_task_pool[t].start_row + 1], 
		// 	   (spmv_task_pool[t].dev_m - 1) * sizeof(int) );

	
		for (int j = 1; j < spmv_task_pool[t].dev_m; j++) {
			spmv_task_pool[t].host_csrRowPtr[j] = (int)(csrRowPtr[spmv_task_pool[t].start_row + j] - spmv_task_pool[t].start_idx);
		}

		spmv_task_pool[t].host_csrColIndex = csrColIndex;
		spmv_task_pool[t].host_csrVal = csrVal;
		spmv_task_pool[t].host_y = y;
		spmv_task_pool[t].host_x = x;

		hipHostMalloc((void **)&(spmv_task_pool[t].local_result_y), spmv_task_pool[t].dev_m * sizeof(double));

		hipHostMalloc((void **)&(spmv_task_pool[t].alpha), 1 * sizeof(double));

		hipHostMalloc((void **)&(spmv_task_pool[t].beta), 1 * sizeof(double));

		spmv_task_pool[t].alpha[0] = *alpha;
		spmv_task_pool[t].beta[0] = *beta;
	}

	for (t = 0; t < num_of_tasks; t++) {
		hipsparseStatus_t status = hipsparseCreateMatDescr(&(spmv_task_pool[t].descr));
		if (status != HIPSPARSE_STATUS_SUCCESS) 
		{ 
			printf("Matrix descriptor initialization failed");
			//return 1;
		} 	
		hipsparseSetMatType(spmv_task_pool[t].descr,HIPSPARSE_MATRIX_TYPE_GENERAL); 
		hipsparseSetMatIndexBase(spmv_task_pool[t].descr,HIPSPARSE_INDEX_BASE_ZERO);
	}

	(*spmv_task_pool_ptr).reserve(num_of_tasks);
	for (t = 0; t < num_of_tasks; t++) {
		(*spmv_task_pool_ptr).push_back(&spmv_task_pool[t]);
	}

}

void assign_task(spmv_task * t, int dev_id, hipStream_t stream){
	t->dev_id = dev_id;
	hipSetDevice(dev_id);

        hipMemcpyAsync(t->dev_csrRowPtr,   t->host_csrRowPtr,          
    			   (size_t)((t->dev_m + 1) * sizeof(int)), hipMemcpyHostToDevice, stream);

	hipMemcpyAsync(t->dev_csrColIndex, &(t->host_csrColIndex[t->start_idx]), 
		           (size_t)(t->dev_nnz * sizeof(int)), hipMemcpyHostToDevice, stream); 

	hipMemcpyAsync(t->dev_csrVal,      &(t->host_csrVal[t->start_idx]),
		           (size_t)(t->dev_nnz * sizeof(double)), hipMemcpyHostToDevice, stream); 

	hipMemcpyAsync(t->dev_y, &(t->host_y[t->start_row]), 
		           (size_t)(t->dev_m * sizeof(double)), hipMemcpyHostToDevice, stream); 
	
	hipMemcpyAsync(t->dev_x, t->host_x,
				   (size_t)(t->dev_n * sizeof(double)),  hipMemcpyHostToDevice, stream);

}

int run_task(spmv_task * t, int dev_id, hipsparseHandle_t handle, int kernel, hipStream_t stream){
	hipSetDevice(dev_id);

	hipsparseStatus_t status = HIPSPARSE_STATUS_SUCCESS;
	int err = 0;
	if(kernel == 1) {
		status = hipsparseDcsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 
								t->dev_m, t->dev_n, t->dev_nnz, 
								t->alpha, t->descr, t->dev_csrVal, 
								t->dev_csrRowPtr, t->dev_csrColIndex, 
								t->dev_x,  t->beta, t->dev_y); 
	} else if (kernel == 2) {
		status = cusparseDcsrmv_mp(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 
									t->dev_m, t->dev_n, t->dev_nnz, 
									t->alpha, t->descr, t->dev_csrVal, 
									t->dev_csrRowPtr, t->dev_csrColIndex, 
									t->dev_x,  t->beta, t->dev_y); 
	} else if (kernel == 3) {
		//cout << "calling csr5_kernel" << endl;
		err = csr5_kernel(t->dev_m, t->dev_n, t->dev_nnz, 
					t->alpha, t->dev_csrVal, 
					t->dev_csrRowPtr, t->dev_csrColIndex, 
					t->dev_x,  t->beta, t->dev_y, stream); 
		//hipDeviceSynchronize();
	}
	if (status != HIPSPARSE_STATUS_SUCCESS || err != 0 ) {
		cout << "KERNEL ERROR: kernel(" <<kernel << ")"<< endl;
		cout << "CSPARSE ERROR: " << status << endl;
		cout << "CSR5 kernel: " << err << endl;
		return -1;
	}
}

void finalize_task(spmv_task * t, int dev_id, hipStream_t stream) {
	hipSetDevice(dev_id);
	hipMemcpyAsync(t->local_result_y,   t->dev_y,          
    			   (size_t)((t->dev_m) * sizeof(double)), 
    			   hipMemcpyDeviceToHost, stream);
	//hipDeviceSynchronize();
}

void gather_results(vector<spmv_task *> * spmv_task_completed, double * y, double * beta, int m) {
	
	int t = 0;
	bool * flag = new bool[m];
	for (int i = 0; i < m; i++) {
		flag[i] = false;
	}
	
	for (t = 0; t < (*spmv_task_completed).size(); t++) {
		 /*
		 cout << "Task " << t << "/" <<(*spmv_task_completed).size() << endl;
		 cout << "flag = " << (*spmv_task_completed)[t]->start_flag <<" " <<   (*spmv_task_completed)[t]->end_flag << endl;
		 for (int i = 0; i < (*spmv_task_completed)[t]->dev_m; i++) {
		 	cout << (*spmv_task_completed)[t]->local_result_y[i] << " ";
		 }
		 cout << endl;
		 */
		double tmp = 0.0;

		if ((*spmv_task_completed)[t]->dev_m == 1 && 
			((*spmv_task_completed)[t]->start_flag) && 
			((*spmv_task_completed)[t]->end_flag)) {
				if (!flag[(*spmv_task_completed)[t]->start_row]) {
					flag[(*spmv_task_completed)[t]->start_row] = true;
				} else {
					tmp = y[(*spmv_task_completed)[t]->start_row];
					(*spmv_task_completed)[t]->local_result_y[0] += tmp;
					(*spmv_task_completed)[t]->local_result_y[0] -= (*beta) * (*spmv_task_completed)[t]->y2;
				}
		}
		
		else {
			if ((*spmv_task_completed)[t]->start_flag) {
				if (!flag[(*spmv_task_completed)[t]->start_row]) {
					flag[(*spmv_task_completed)[t]->start_row] = true;
				} else {
					tmp = y[(*spmv_task_completed)[t]->start_row];
					(*spmv_task_completed)[t]->local_result_y[0] += tmp;
					(*spmv_task_completed)[t]->local_result_y[0] -= (*beta) * (*spmv_task_completed)[t]->y2;
				}
			}

			if ((*spmv_task_completed)[t]->end_flag) {
				if (!flag[(*spmv_task_completed)[t]->end_row]) {
					flag[(*spmv_task_completed)[t]->end_row] = true;
				} else {
					tmp = y[(*spmv_task_completed)[t]->end_row];
					(*spmv_task_completed)[t]->local_result_y[(*spmv_task_completed)[t]->dev_m - 1] += tmp;
					(*spmv_task_completed)[t]->local_result_y[(*spmv_task_completed)[t]->dev_m - 1] -= (*beta) * (*spmv_task_completed)[t]->y2;
				}
			}
		}

		memcpy(&y[(*spmv_task_completed)[t]->start_row], 
			   (*spmv_task_completed)[t]->local_result_y, 
			  ((*spmv_task_completed)[t]->dev_m * sizeof(double))); 

	}
}

void print_task_info(spmv_task * t) {
	cout << "start_idx = " << t->start_idx << endl;
	cout << "end_idx = " << t->end_idx << endl;
}
