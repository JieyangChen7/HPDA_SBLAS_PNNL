#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <ctime>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "hipsparse.h"
#include <iostream>
#include <cmath>
#include <float.h>
//#include "anonymouslib_cuda.h"
#include "spmv_kernel.h"
using namespace std;


spmv_ret spMV_mgpu_baseline_coo(int m, int n, int nnz, double * alpha,
            double * cooVal, int * cooRowIdx, int * cooColIdx, 
            double * x, double * beta,
            double * y,
            int ngpu){

  double curr_time = 0.0;
  double numa_part_time = 0.0;
  double part_time = 0.0;
  double comp_time = 0.0;
  double comm_time = 0.0;
  double merg_time = 0.0;

  hipStream_t * stream = new hipStream_t [ngpu];
  hipsparseHandle_t * handle = new hipsparseHandle_t[ngpu];
  hipsparseMatDescr_t * descr = new hipsparseMatDescr_t[ngpu];

  int  * start_row  = new int[ngpu];
  int  * end_row    = new int[ngpu];

  int  * start_idx  = new int[ngpu];
  int  * end_idx    = new int[ngpu];
    
  int * dev_m            = new int      [ngpu];
  int * dev_n            = new int      [ngpu];
  int * dev_nnz          = new int      [ngpu];
  
  int ** host_cooRowIdx  = new int    * [ngpu];

  int ** dev_cooRowIdx   = new int    * [ngpu];
  int ** dev_cooColIdx   = new int    * [ngpu];
  double ** dev_cooVal   = new double * [ngpu];

  int ** dev_csrRowPtr   = new int    * [ngpu];
  int ** dev_csrColIdx   = new int    * [ngpu];
  double ** dev_csrVal   = new double * [ngpu];

  double ** dev_x = new double * [ngpu];
  double ** dev_y = new double * [ngpu];

  double ** host_py = new double * [ngpu];

  curr_time = get_time();
  for (int d = 0; d < ngpu; d++) {
    checkCudaErrors(hipSetDevice(d));
    start_row[d] = floor((d)     * m / ngpu);
    end_row[d]   = floor((d + 1) * m / ngpu) - 1;
    start_idx[d] = findFirstInSorted(cooRowIdx, nnz, start_row[d]);
    end_idx[d]   = findLastInSorted(cooRowIdx, nnz, end_row[d]);
    dev_m[d]   = end_row[d] - start_row[d] + 1;
    dev_n[d]   = n;
    dev_nnz[d] = end_idx[d] - start_idx[d] + 1;
  }
  part_time += get_time() - curr_time;

  for (int d = 0; d < ngpu; d++) {
    checkCudaErrors(hipSetDevice(d));
    checkCudaErrors(hipStreamCreate(&(stream[d])));
    checkCudaErrors(hipsparseCreate(&(handle[d]))); 
    checkCudaErrors(hipsparseSetStream(handle[d], stream[d]));
    checkCudaErrors(hipsparseCreateMatDescr(&descr[d]));
    checkCudaErrors(hipsparseSetMatType(descr[d],HIPSPARSE_MATRIX_TYPE_GENERAL)); 
    checkCudaErrors(hipsparseSetMatIndexBase(descr[d],HIPSPARSE_INDEX_BASE_ZERO)); 

    checkCudaErrors(hipMalloc((void**)&(dev_csrVal[d]),      dev_nnz[d]     * sizeof(double)));
    checkCudaErrors(hipMalloc((void**)&(dev_csrRowPtr[d]),   (dev_m[d] + 1) * sizeof(int)   ));
    checkCudaErrors(hipMalloc((void**)&(dev_csrColIdx[d]),   dev_nnz[d]     * sizeof(int)   ));

    checkCudaErrors(hipHostMalloc((void**)& (host_py[d]), dev_m[d] * sizeof(double)));
    checkCudaErrors(hipHostMalloc((void**)& (host_cooRowIdx[d]), dev_nnz[d] * sizeof(int)));

    checkCudaErrors(hipMalloc((void**)&dev_cooVal[d],    dev_nnz[d] * sizeof(double)));
    checkCudaErrors(hipMalloc((void**)&dev_cooRowIdx[d], dev_nnz[d] * sizeof(int))); 
    checkCudaErrors(hipMalloc((void**)&dev_cooColIdx[d], dev_nnz[d] * sizeof(int))); 

    checkCudaErrors(hipMalloc((void**)&dev_x[d],           dev_n[d] * sizeof(double))); 
    checkCudaErrors(hipMalloc((void**)&dev_y[d],           dev_m[d] * sizeof(double))); 

  }

  curr_time = get_time();
  for (int d = 0; d < ngpu; d++) {
    for (int i = 0; i < dev_nnz[d]; i++) {
      host_cooRowIdx[d][i] = cooRowIdx[start_idx[d]] - start_row[d];
    }
  }
  part_time += get_time() - curr_time;

  curr_time = get_time();
  for (int d = 0; d < ngpu; d++) {
    checkCudaErrors(hipSetDevice(d));
    checkCudaErrors(hipMemcpyAsync(dev_cooVal[d],    &(cooVal[start_idx[d]]),          dev_nnz[d] * sizeof(double), hipMemcpyHostToDevice, stream[d]));
    checkCudaErrors(hipMemcpyAsync(dev_cooRowIdx[d], &host_cooRowIdx[start_idx[d]], dev_nnz[d] * sizeof(int),    hipMemcpyHostToDevice, stream[d])); 
    checkCudaErrors(hipMemcpyAsync(dev_cooColIdx[d], &cooColIdx[start_idx[d]],      dev_nnz[d] * sizeof(int), hipMemcpyHostToDevice, stream[d]));
    checkCudaErrors(hipMemcpyAsync(dev_y[d],         &y[start_row[d]],                 dev_m[d]*sizeof(double),     hipMemcpyHostToDevice, stream[d])); 
    checkCudaErrors(hipMemcpyAsync(dev_x[d],         x,                             dev_n[d]*sizeof(double),     hipMemcpyHostToDevice, stream[d])); 
  }
  //time_comm = get_time() - curr_time;


  //curr_time = get_time();
  for (int d = 0; d < ngpu; ++d) {
    checkCudaErrors(hipSetDevice(d));
    coo2csr_gpu(handle[d], stream[d], dev_m[d], dev_n[d], dev_nnz[d],
                dev_cooVal[d], dev_cooRowIdx[d], dev_cooColIdx[d],
                dev_csrVal[d], dev_csrRowPtr[d], dev_csrColIdx[d]);
    // checkCudaErrors(hipsparseDcsrmv(handle[d],HIPSPARSE_OPERATION_NON_TRANSPOSE, 
    //                            dev_m[d], dev_n[d], dev_nnz[d], 
    //                            alpha, descr[d], dev_csrVal[d], 
    //                            dev_csrRowPtr[d], dev_csrColIdx[d], 
    //                            dev_x[d], beta, dev_y[d]));       
  }
  for (int d = 0; d < ngpu; ++d) {
    checkCudaErrors(hipSetDevice(d));
    checkCudaErrors(hipDeviceSynchronize());
  }
  comp_time = get_time() - curr_time;

  curr_time = get_time();
  for (int d = 0; d < ngpu; d++) {
    checkCudaErrors(hipSetDevice(d));
    checkCudaErrors(hipMemcpyAsync(&y[start_row[d]],   dev_y[d], dev_m[d]*sizeof(double),     hipMemcpyDeviceToHost, stream[d])); 
    //checkCudaErrors(hipMemcpyAsync(host_py[d], dev_y[d], 
                    //dev_m[d] * sizeof(double), hipMemcpyDeviceToHost, stream[d])); 
  }
  for (int d = 0; d < ngpu; d++) {
    checkCudaErrors(hipSetDevice(d));
    checkCudaErrors(hipDeviceSynchronize());
  }
  // for (int d = 0; d < ngpu; d++) {
  //   for (int i = 0; i < dev_m[d]; i++) {
  //     y[start_row[d] + i] += host_py[d][i];
  //   }
  // }
  merg_time = get_time() - curr_time;

  for (int d = 0; d < ngpu; d++) {
    hipFree(dev_csrVal[d]);
    hipFree(dev_csrRowPtr[d]);
    hipFree(dev_csrColIdx[d]);
    hipHostFree(host_py[d]);
    hipHostFree(host_cooRowIdx[d]);

    hipFree(dev_cooColIdx[d]);
    hipFree(dev_cooRowIdx[d]);
    hipFree(dev_cooVal[d]);

    hipFree(dev_x[d]);
    hipFree(dev_y[d]);

    hipsparseDestroyMatDescr(descr[d]);
    hipsparseDestroy(handle[d]);
    hipStreamDestroy(stream[d]);
  }

  delete [] stream;
  delete [] handle;
  delete [] descr;
  delete [] start_row;
  delete [] end_row;
  delete [] dev_m;
  delete [] dev_n;
  delete [] dev_nnz;
  delete [] host_cooRowIdx;
  delete [] dev_cooColIdx;
  delete [] dev_cooRowIdx;
  delete [] dev_cooVal;
  delete [] dev_csrRowPtr;
  delete [] dev_csrColIdx;
  delete [] dev_csrVal;
  delete [] dev_x;
  delete [] dev_y;
  delete [] host_py;
    
  spmv_ret ret;
  ret.numa_part_time = numa_part_time;
  ret.part_time = part_time;
  ret.comp_time = comp_time;
  ret.comm_time = comm_time;
  ret.merg_time = merg_time;
  return ret;

}
