#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <ctime>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "hipsparse.h"
#include <iostream>
#include <cmath>
#include <float.h>
//#include "anonymouslib_cuda.h"
#include "spmv_kernel.h"
#include <vector>
#include <hip/hip_runtime_api.h>
#include <omp.h>
#include <sched.h>
#include <hip/library_types.h>
using namespace std;


spmv_ret spMV_mgpu_v1_numa_csc(int m, int n, long long nnz, double * alpha,
          double * cscVal, long long * cscColPtr, int * cscRowIndex, 
          double * x, double * beta,
          double * y,
          int ngpu, 
          int kernel,
          int * numa_mapping){

  double curr_time = 0.0;
  double time_comm = 0.0;
  double time_comp = 0.0;


  struct NumaContext numaContext(numa_mapping, ngpu);
  struct pCSC * pcscNuma = new struct pCSC[numaContext.numNumaNodes];

  double numa_part_time;
  
  omp_set_num_threads(ngpu);
  #pragma omp parallel default (shared) reduction(max:numa_part_time)
  {
    string s;
    unsigned int dev_id = omp_get_thread_num();
    unsigned int hwthread = sched_getcpu();

    int numa_id = numaContext.numaMapping[dev_id];
  
    numa_part_time = 0;

    if (numaContext.representiveThreads[dev_id]) {
      printf("represent thread %d hw thread %d\n", dev_id, hwthread);

      double tmp_time = get_time();

      int tmp1 = numaContext.workload[numa_id] * nnz;
      int tmp2 = numaContext.workload[numa_id + 1] * nnz;

      pcscNuma[numa_id].startIdx = floor((double)tmp1 / ngpu);
      pcscNuma[numa_id].endIdx = floor((double)tmp2 / ngpu) - 1;


      // Calculate the start and end row
      pcscNuma[numa_id].startCol = get_row_from_index(n+1, cscColPtr, pcscNuma[numa_id].startIdx);
      // Mark imcomplete rows
      // True: imcomplete
      if (pcscNuma[numa_id].startIdx > cscColPtr[pcscNuma[numa_id].startCol]) {
        pcscNuma[numa_id].startFlag = true;
      } else {
        pcscNuma[numa_id].startFlag = false;
      }

      pcscNuma[numa_id].endCol = get_row_from_index(n+1, cscColPtr, pcscNuma[numa_id].endIdx);
      // Mark imcomplete rows
      // True: imcomplete
      if (pcscNuma[numa_id].endIdx < cscColPtr[pcscNuma[numa_id].endCol + 1] - 1)  {
        pcscNuma[numa_id].endFlag = true;

      } else {
        pcscNuma[numa_id].endFlag = false;

      }

      // Cacluclate dimensions
      pcscNuma[numa_id].m = m;
      pcscNuma[numa_id].n = pcscNuma[numa_id].endCol - pcscNuma[numa_id].startCol + 1;
      pcscNuma[numa_id].nnz  = pcscNuma[numa_id].endIdx - pcscNuma[numa_id].startIdx + 1;

      printf("numa_id %d, numa_start_idx %d, numa_end_idx %d\n",numa_id, pcscNuma[numa_id].startIdx, pcscNuma[numa_id].endIdx);
      printf("numa_id %d, numa_start_row %d, numa_end_row %d\n",numa_id, pcscNuma[numa_id].startCol, pcscNuma[numa_id].endCol);
    
      numa_part_time += get_time() - tmp_time;

      // preparing data on host 
      hipHostMalloc((void**)&(pcscNuma[numa_id].val), pcscNuma[numa_id].nnz * sizeof(double));
      hipHostMalloc((void**)&(pcscNuma[numa_id].colPtr), (pcscNuma[numa_id].n + 1)*sizeof(int));
      hipHostMalloc((void**)&(pcscNuma[numa_id].rowIdx), pcscNuma[numa_id].nnz * sizeof(int));
      hipHostMalloc((void**)&(pcscNuma[numa_id].x), pcscNuma[numa_id].n * sizeof(double));
      hipHostMalloc((void**)&(pcscNuma[numa_id].y), pcscNuma[numa_id].m * sizeof(double));

      tmp_time = get_time();

      for (int i = pcscNuma[numa_id].startIdx; i <= pcscNuma[numa_id].endIdx; i++) {
        pcscNuma[numa_id].val[i - pcscNuma[numa_id].startIdx] = cscVal[i];
      }

      // for (int i = numa_start_idx[numa_id]; i <= numa_end_idx[numa_id]; i++) {
      //   numa_csrVal[numa_id][i - numa_start_idx[numa_id]] = csrVal[i];
      // }


      pcscNuma[numa_id].colPtr[0] = 0;
      pcscNuma[numa_id].colPtr[pcscNuma[numa_id].n] = pcscNuma[numa_id].nnz;
      for (int j = 1; j < pcscNuma[numa_id].n; j++) {
        pcscNuma[numa_id].colPtr[j] = pcscNuma[pcscNuma[numa_id].startCol + j] - pcscNuma[numa_id].startIdx;
      }

      for (int i = pcscNuma[numa_id].startIdx; i <= pcscNuma[numa_id].endIdx; i++) {
        pcscNuma[numa_id].rowIdx[i - pcscNuma[numa_id].startIdx] = cscRowIndex[i];
      }

      for (int i = 0; i < pcscNuma[numa_id].n; i++) {
        pcscNuma[numa_id].x[i] = x[pcscNuma[numa_id].startCol + i];
      }
    
      for (int i = 0; i < pcscNuma[numa_id].m; i++) {
        pcscNuma[numa_id].y[i] = y[i];
      }

      numa_part_time += get_time() - tmp_time;

    }
  }




  
  // //printf("test0\n");
  // double * start_element = new double[ngpu];
  // double * end_element = new double[ngpu];
  // bool * start_flags = new bool[ngpu];
  // bool * end_flags = new bool[ngpu];
  // double * org_y = new double[ngpu];
  // int * start_rows = new int[ngpu];
  // //printf("test01\n");

  // omp_set_num_threads(ngpu);

  // double core_time;
  // double part_time;
  // double merg_time;
  // #pragma omp parallel default (shared) reduction(max:core_time) reduction(max:part_time) reduction(max:merg_time)
  // {
  //   unsigned int dev_id = omp_get_thread_num();
  //   hipSetDevice(dev_id);
  //   unsigned int hwthread = sched_getcpu();

  //   printf("omp thread %d, hw thread %d\n", dev_id, hwthread);  

  //   long long  start_idx, end_idx;
  //   int start_col, end_col;
  //   bool start_flag, end_flag; 


  //   double * host_cscVal;
  //   int    * host_cscColPtr;
  //   int    * host_cscRowIndex;
  //   double * host_x;
  //   double * host_y;

  //   double * dev_cscVal;
  //   int    * dev_cscColPtr;
  //   int    * dev_cscRowIndex;

  //   int    dev_nnz, dev_m, dev_n;

    
  //   double * dev_x;
  //   double * dev_y;
  //   double y2;

  //   hipStream_t stream;
  //   hipsparseStatus_t status;
  //   hipsparseHandle_t handle;
  //   hipsparseMatDescr_t descr;
  //   int err;


  //   double tmp_time = get_time();

  //   // Calculate the start and end index
  //   long long tmp1 = dev_id * nnz;
  //   long long tmp2 = (dev_id + 1) * nnz;

  //   //double tmp3 = (double)(tmp1 / ngpu);
  //   //double tmp4 = (double)(tmp2 / ngpu);

  //   start_idx = floor((double)tmp1 / ngpu);
  //   end_idx   = floor((double)tmp2 / ngpu) - 1;
  
  //   // Calculate the start and end col
  //   start_col = get_row_from_index(m, cscColPtr, start_idx);
  //   // Mark imcomplete rows
  //   // True: imcomplete
  //   if (start_idx > cscColPtr[start_col]) {
  //     start_flag = true;
  //     start_rows[dev_id] = start_row;
  //   } else {
  //     start_flag = false;
  //   }
  //   start_flags[dev_id] = start_flag;   

  //   end_col = get_row_from_index(m, cscColPtr, end_idx);
  //   // Mark imcomplete rows
  //   // True: imcomplete
  //   if (end_idx < cscColPtr[end_row + 1] - 1)  {
  //     end_flag = true;
  //   } else {
  //     end_flag = false;
  //   }
    
  //   // Cacluclate dimensions
  //   dev_m = m;
  //   dev_n = end_col - start_col + 1;
  //   dev_nnz   = (int)(end_idx - start_idx + 1);

  //   part_time = get_time() - tmp_time;  



  //   // preparing data on host 
  //   //hipHostMalloc((void**)&host_csrVal, dev_nnz * sizeof(double));
  //   //for (int i = start_idx; i <= end_idx; i++) {
  //   //  host_csrVal[i - start_idx] = csrVal[i];
  //   //}

  //   hipHostMalloc((void**)&host_cscColPtr, (dev_n + 1)*sizeof(int));
  //   //host_csrRowPtr[0] = 0;
  //   //host_csrRowPtr[dev_m] = dev_nnz;
  //   //for (int j = 1; j < dev_m; j++) {
  //   //  host_csrRowPtr[j] = (int)(csrRowPtr[start_row + j] - start_idx);
  //   //}

  //   //hipHostMalloc((void**)&host_csrColIndex, dev_nnz * sizeof(int));
  //   //for (int i = start_idx; i <= end_idx; i++) {
  //   //  host_csrColIndex[i - start_idx] = csrColIndex[i];
  //   //}

  //   //hipHostMalloc((void**)&host_x, dev_n * sizeof(double));
  //   //for (int i = 0; i < dev_n; i++) {
  //   //  host_x[i] = x[i];
  //   //}

  //   //hipHostMalloc((void**)&host_y, dev_m * sizeof(double));
  //   //for (int i = 0; i < dev_m; i++) {
  //   //  host_y[i] = y[start_row + i];
  //   //}
    
  //   tmp_time = get_time();

  //   host_cscVal = cscVal[start_idx];
  //   host_cscColPtr[0] = 0;
  //   host_cscColPtr[dev_n] = dev_nnz;
  //   for (int j = 1; j < dev_m; j++) {
  //     host_cscColPtr[j] = (int)(cscColPtr[start_col + j] - start_idx);
  //   }
  //   host_cscRowIndex = cscRowIndex[start_idx];
  //   host_x = &x[start_col];
  //   host_y = y;
  
  //   part_time += get_time() - tmp_time;

  


    



  //   // preparing GPU env
  //   hipStreamCreate(&stream);

  //   status = hipsparseCreate(&handle); 
  //   if (status != HIPSPARSE_STATUS_SUCCESS) 
  //   { 
  //     printf("CUSPARSE Library initialization failed");
  //     //return 1; 
  //   } 
  //   status = hipsparseSetStream(handle, stream);
  //   if (status != HIPSPARSE_STATUS_SUCCESS) 
  //   { 
  //     printf("Stream bindind failed");
  //     //return 1;
  //   } 
  //   status = hipsparseCreateMatDescr(&descr);
  //   if (status != HIPSPARSE_STATUS_SUCCESS) 
  //   { 
  //     printf("Matrix descriptor initialization failed");
  //     //return 1;
  //   }   
  //   hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL); 
  //   hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO);
    
  //   hipMalloc((void**)&dev_cscVal,      dev_nnz     * sizeof(double));
  //   hipMalloc((void**)&dev_cscColPtr,   (dev_n + 1) * sizeof(int)   );
  //   hipMalloc((void**)&dev_cscRowIndex, dev_nnz     * sizeof(int)   );
  //   hipMalloc((void**)&dev_x,           dev_n       * sizeof(double)); 
  //   hipMalloc((void**)&dev_y,           dev_m       * sizeof(double)); 

  //   double * dense_A;
  //   double lda = dev_m;
  //   hipMalloc((void**)&dense_A, dev_n * dev_m * sizeof(double));

  //         //hipProfilerStart();
  //   #pragma omp barrier
  //   tmp_time = get_time();

  //   hipMemcpyAsync(dev_cscColPtr, host_cscColPtr, (dev_n + 1) * sizeof(int), hipMemcpyHostToDevice, stream);
  //   hipMemcpyAsync(dev_cscRowIndex, host_cscRowIndex, dev_nnz * sizeof(int), hipMemcpyHostToDevice, stream); 
  //   hipMemcpyAsync(dev_cscVal, host_cscVal, dev_nnz * sizeof(double), hipMemcpyHostToDevice, stream); 
  //   hipMemcpyAsync(dev_y, host_y, dev_m*sizeof(double),  hipMemcpyHostToDevice, stream); 
  //   hipMemcpyAsync(dev_x, host_x, dev_n*sizeof(double), hipMemcpyHostToDevice, stream); 
    
    
  //   time_comm = get_time() - curr_time;
  //   curr_time = get_time();

  //   err = 0;
  //   if (kernel == 1) {

  //     hipsparseDcsc2dense(handle,
  //                        dev_m, dev_n,
  //                        descr,
  //                        dev_cscVal,
  //                        dev_cscRowIndex,
  //                        dev_cscColPtr,
  //                        dense_A,
  //                        lda);

      


  //     status = hipsparseDcsrmv(handle,HIPSPARSE_OPERATION_NON_TRANSPOSE, 
  //                             dev_m, dev_n, dev_nnz, 
  //                             alpha, descr, dev_csrVal, 
  //                             dev_csrRowPtr, dev_csrColIndex, 
  //                             dev_x, beta, dev_y);
      
  //     /*
  //     hipsparseSpMatDescr_t A_desc;
  //     hipsparseCreateCsr(&A_desc, dev_m, dev_n, dev_nnz, 
  //                       dev_csrVal, dev_csrRowPtr, dev_csrColIndex,
  //                       HIPSPARSE_INDEX_32I, 
  //                       HIPSPARSE_INDEX_32I, 
  //                       HIPSPARSE_INDEX_BASE_ZERO, 
  //                       HIP_R_64F);
  //     hipsparseDnVecDescr_t x_desc;
  //     hipsparseCreateDnVec(&x_desc, dev_n, dev_x, HIP_R_64F);
  
  //     hipsparseDnVecDescr_t y_desc;
  //     hipsparseCreateDnVec(&y_desc, dev_m, dev_y, HIP_R_64F);

  //     size_t buffer_size;
  //     hipsparseSpMV_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
  //                             &alpha, A_desc, x_desc, &beta, y_desc,
  //                             HIP_R_64F, HIPSPARSE_MV_ALG_DEFAULT, 
  //                             &buffer_size);
  //     void * buffer;
  //     hipMalloc(&buffer, buffer_size);

  //     hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
  //                  &alpha, A_desc, x_desc, &beta, y_desc,
  //                  HIP_R_64F, HIPSPARSE_MV_ALG_DEFAULT, 
  //                  buffer);

      
  //     */
  
 
  //   } else if (kernel == 2) {
  //       status = cusparseDcsrmv_mp(handle,HIPSPARSE_OPERATION_NON_TRANSPOSE, 
  //                                  dev_m, dev_n, dev_nnz, 
  //                                  alpha, descr, dev_csrVal, 
  //                                  dev_csrRowPtr, dev_csrColIndex, 
  //                                  dev_x,  beta, dev_y); 
  //   } else if (kernel == 3) {
  //       err = csr5_kernel(dev_m, dev_n, dev_nnz, 
  //                         alpha, dev_csrVal, 
  //                         dev_csrRowPtr, dev_csrColIndex, 
  //                         dev_x, beta, dev_y, stream); 
  //   }

  //   core_time = get_time() - tmp_time;
  //   // GPU based merge
  //   tmp_time = get_time();
  //   double * dev_y_no_overlap = dev_y;
  //   int dev_m_no_overlap = dev_m;
  //   int start_row_no_overlap = start_row;
  //   if (start_flag) {
  //     dev_y_no_overlap += 1;
  //     start_row_no_overlap += 1;
  //     dev_m_no_overlap -= 1;
  //     hipMemcpyAsync(start_element+dev_id, dev_y, sizeof(double), hipMemcpyDeviceToHost, stream);
  //   }
  //   hipMemcpyAsync(y+start_row_no_overlap, dev_y_no_overlap, dev_m_no_overlap*sizeof(double),  hipMemcpyDeviceToHost, stream);
  //   hipDeviceSynchronize();
  //   #pragma omp barrier
  //   if (dev_id == 0) {
  //     for (int i = 0; i < ngpu; i++) {
  //       if (start_flags[i]) {
  //         y[start_rows[i]] += (start_element[i] - (*beta) * org_y[i]); 
  //       } 
  //     }
  //   }

  //   /* CPU based merge
  //   hipMemcpyAsync(host_y, dev_y, dev_m*sizeof(double),  hipMemcpyDeviceToHost, stream);

  //   hipDeviceSynchronize();
    
    
  //   //printf("thread %d time: %f\n", dev_id,  get_time() - tmp_time);
  //   #pragma omp critical
  //   {
  //     double tmp = 0.0;
      
  //     if (start_flag) {
  //       tmp = y[start_row];
  //     }

  //     for (int i = 0; i < dev_m; i++) y[start_row + i] = host_y[i];

  //     if (start_flag) {
  //       y[start_row] += tmp;
  //       y[start_row] -= y2 * (*beta);
  //     }
  //   }
  //   */
  //   merg_time = get_time() - tmp_time;

  //   //hipProfilerStop();

  //   hipFree(dev_csrVal);
  //   hipFree(dev_csrRowPtr);
  //   hipFree(dev_csrColIndex);
  //   hipFree(dev_x);
  //   hipFree(dev_y);
          
  //   hipHostFree(host_csrRowPtr);
  //   hipHostFree(host_csrVal);
  //   hipHostFree(host_csrColIndex);
  //   hipHostFree(host_x);
  //   hipHostFree(host_y);

  //   hipsparseDestroyMatDescr(descr);
  //   hipsparseDestroy(handle);
  //   hipStreamDestroy(stream);

  //   }

    //cout << "time_parse = " << time_parse << ", time_comm = " << time_comm << ", time_comp = "<< time_comp <<", time_post = " << time_post << endl;
                spmv_ret ret;
                ret.comp_time = core_time;
                ret.comm_time = 0.0;
                ret.part_time = part_time;
                ret.merg_time = merg_time;
    return ret;
  }

