#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <ctime>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "hipsparse.h"
#include <iostream>
#include <cmath>
#include <float.h>
//#include "anonymouslib_cuda.h"
#include "spmv_kernel.h"
#include <vector>
#include <hip/hip_runtime_api.h>
#include <omp.h>
#include <sched.h>
#include <hip/library_types.h>
#include <string>
using namespace std;

__global__ void
_calcCscColPtr(int * cscColPrt, int m, int offset, int nnz) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  for (int i = idx; i < m; i += blockDim.x * gridDim.x) {
    cscColPrt[i] -= offset; 
    //printf("thread %d: %d - %d\n", idx, csrRowPrt[i], offset);
  }
  if (idx == 0) {
    cscColPrt[0] = 0;
    cscColPrt[m] = nnz;
  }
}

void calcCscColPtr(int * cscColPrt, int m, int offset, int nnz, hipStream_t stream) {
  int thread_per_block = 256;
  int block_per_grid = ceil((float)m / thread_per_block); 
  _calcCscColPtr<<<block_per_grid, thread_per_block, 0, stream>>>(cscColPrt, m, offset, nnz);
}





spmv_ret spMV_mgpu_v1_numa_csc(int m, int n, long long nnz, double * alpha,
          double * cscVal, int * cscColPtr, int * cscRowIndex, 
          double * x, double * beta,
          double * y,
          int ngpu, 
          int kernel,
          int * numa_mapping,
          int part_opt,
          int merg_opt){

  double numa_part_time = 0.0;
  double part_time = 0.0;
  double comp_time = 0.0;
  double comm_time = 0.0;
  double merg_time = 0.0;


  struct NumaContext numaContext(numa_mapping, ngpu);
  struct pCSC * pcscNuma = new struct pCSC[numaContext.numNumaNodes];

  omp_set_num_threads(ngpu);
  #pragma omp parallel default (shared) reduction(max:numa_part_time)
  {
    string s;
    unsigned int dev_id = omp_get_thread_num();
    unsigned int hwthread = sched_getcpu();

    int numa_id = numaContext.numaMapping[dev_id];
  
    numa_part_time = 0;

    if (numaContext.representiveThreads[dev_id]) {
      // printf("represent thread %d hw thread %d\n", dev_id, hwthread);

      double tmp_time = get_time();

      int tmp1 = numaContext.workload[numa_id] * nnz;
      int tmp2 = numaContext.workload[numa_id + 1] * nnz;

      pcscNuma[numa_id].startIdx = floor((double)tmp1 / ngpu);
      pcscNuma[numa_id].endIdx = floor((double)tmp2 / ngpu) - 1;


      // Calculate the start and end row
      pcscNuma[numa_id].startCol = get_row_from_index(n+1, cscColPtr, pcscNuma[numa_id].startIdx);
      // Mark imcomplete rows
      // True: imcomplete
      if (pcscNuma[numa_id].startIdx > cscColPtr[pcscNuma[numa_id].startCol]) {
        pcscNuma[numa_id].startFlag = true;
      } else {
        pcscNuma[numa_id].startFlag = false;
      }

      pcscNuma[numa_id].endCol = get_row_from_index(n+1, cscColPtr, pcscNuma[numa_id].endIdx);
      // Mark imcomplete rows
      // True: imcomplete
      if (pcscNuma[numa_id].endIdx < cscColPtr[pcscNuma[numa_id].endCol + 1] - 1)  {
        pcscNuma[numa_id].endFlag = true;

      } else {
        pcscNuma[numa_id].endFlag = false;

      }

      // Cacluclate dimensions
      pcscNuma[numa_id].m = m;
      pcscNuma[numa_id].n = pcscNuma[numa_id].endCol - pcscNuma[numa_id].startCol + 1;
      pcscNuma[numa_id].nnz  = pcscNuma[numa_id].endIdx - pcscNuma[numa_id].startIdx + 1;

      // // printf("numa_id %d, numa_start_idx %d, numa_end_idx %d\n",
      //         numa_id, pcscNuma[numa_id].startIdx, pcscNuma[numa_id].endIdx);
      // printf("numa_id %d, numa_start_col %d, numa_end_col %d\n",
      //         numa_id, pcscNuma[numa_id].startCol, pcscNuma[numa_id].endCol);
    
      numa_part_time += get_time() - tmp_time;

      // preparing data on host 
      hipHostMalloc((void**)&(pcscNuma[numa_id].val), pcscNuma[numa_id].nnz * sizeof(double));
      hipHostMalloc((void**)&(pcscNuma[numa_id].colPtr), (pcscNuma[numa_id].n + 1)*sizeof(int));
      hipHostMalloc((void**)&(pcscNuma[numa_id].rowIdx), pcscNuma[numa_id].nnz * sizeof(int));
      hipHostMalloc((void**)&(pcscNuma[numa_id].x), pcscNuma[numa_id].n * sizeof(double));
      hipHostMalloc((void**)&(pcscNuma[numa_id].y), pcscNuma[numa_id].m * sizeof(double));

      tmp_time = get_time();

      for (int i = pcscNuma[numa_id].startIdx; i <= pcscNuma[numa_id].endIdx; i++) {
        pcscNuma[numa_id].val[i - pcscNuma[numa_id].startIdx] = cscVal[i];
      }

      // for (int i = numa_start_idx[numa_id]; i <= numa_end_idx[numa_id]; i++) {
      //   numa_csrVal[numa_id][i - numa_start_idx[numa_id]] = csrVal[i];
      // }


      pcscNuma[numa_id].colPtr[0] = 0;
      pcscNuma[numa_id].colPtr[pcscNuma[numa_id].n] = pcscNuma[numa_id].nnz;
      for (int j = 1; j < pcscNuma[numa_id].n; j++) {
        pcscNuma[numa_id].colPtr[j] = cscColPtr[pcscNuma[numa_id].startCol + j] - pcscNuma[numa_id].startIdx;
      }

      for (int i = pcscNuma[numa_id].startIdx; i <= pcscNuma[numa_id].endIdx; i++) {
        pcscNuma[numa_id].rowIdx[i - pcscNuma[numa_id].startIdx] = cscRowIndex[i];
      }

      for (int i = 0; i < pcscNuma[numa_id].n; i++) {
        pcscNuma[numa_id].x[i] = x[pcscNuma[numa_id].startCol + i];
      }
    
      for (int i = 0; i < pcscNuma[numa_id].m; i++) {
        pcscNuma[numa_id].y[i] = y[i];
      }

      numa_part_time += get_time() - tmp_time;

      // print_vec(pcscNuma[numa_id].val, pcscNuma[numa_id].nnz, "cscVal"+to_string(dev_id));
      // print_vec(pcscNuma[numa_id].colPtr, pcscNuma[numa_id].n + 1, "colPtr"+to_string(dev_id));
      // print_vec(pcscNuma[numa_id].rowIdx, pcscNuma[numa_id].nnz, "rowIdx"+to_string(dev_id));
      // print_vec(pcscNuma[numa_id].x, pcscNuma[numa_id].n, "x"+to_string(dev_id));
      // print_vec(pcscNuma[numa_id].y, pcscNuma[numa_id].m, "y_before"+to_string(dev_id));
      // printf("dev_id %d, alpha %f, beta %f\n", dev_id, *alpha, *beta);

    }
  }


  struct pCSC * pcscGPU = new struct pCSC[ngpu];

  double * start_element = new double[ngpu];
  double * end_element = new double[ngpu];
  bool * start_flags = new bool[ngpu];
  bool * end_flags = new bool[ngpu];
  double * org_y = new double[ngpu];
  int * start_rows = new int[ngpu];

  omp_set_num_threads(ngpu);
  #pragma omp parallel default (shared) reduction(max:comp_time) reduction(max:part_time) reduction(max:merg_time)
  {
    unsigned int dev_id = omp_get_thread_num();
    hipSetDevice(dev_id);
    unsigned int hwthread = sched_getcpu();

    float elapsedTime;

    hipStream_t stream;
    hipsparseStatus_t status;
    hipsparseHandle_t handle;
    hipsparseMatDescr_t descr;

    checkCudaErrors(hipStreamCreate(&stream));
    checkCudaErrors(hipsparseCreate(&handle)); 
    checkCudaErrors(hipsparseSetStream(handle, stream));
    checkCudaErrors(hipsparseCreateMatDescr(&descr));
    checkCudaErrors(hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL)); 
    checkCudaErrors(hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO));

    hipEvent_t comp_start, comp_stop;
    hipEvent_t comm_start, comm_stop;

    checkCudaErrors(hipEventCreate(&comp_start));
    checkCudaErrors(hipEventCreate(&comp_stop));
    checkCudaErrors(hipEventCreate(&comm_start));
    checkCudaErrors(hipEventCreate(&comm_stop));


    // printf("omp thread %d, hw thread %d\n", dev_id, hwthread);  

    int numa_id = numaContext.numaMapping[dev_id];
    int local_dev_id = 0;
    for (int i = 0; i < ngpu; i++) {
      if (i == dev_id) break;
      if (numa_id == numaContext.numaMapping[i]) local_dev_id++;
    }
  
    double tmp_time = get_time();

    // Calculate the start and end index
    long long tmp1 = local_dev_id * pcscNuma[numa_id].nnz;
    long long tmp2 = (local_dev_id + 1) * pcscNuma[numa_id].nnz;

    pcscGPU[dev_id].startIdx = floor((double)tmp1 / numaContext.numGPUs[numa_id]);
    pcscGPU[dev_id].endIdx   = floor((double)tmp2 / numaContext.numGPUs[numa_id]) - 1;
  
    // Calculate the start and end col
    pcscGPU[dev_id].startCol = get_row_from_index(pcscNuma[numa_id].n, pcscNuma[numa_id].colPtr, pcscGPU[dev_id].startIdx);
    // Mark imcomplete rows
    // True: imcomplete
    if (pcscGPU[dev_id].startIdx > pcscNuma[numa_id].colPtr[pcscGPU[dev_id].startCol]) {
      pcscGPU[dev_id].startFlag = true;
      //start_rows[dev_id] = start_row;
    } else {
      pcscGPU[dev_id].startFlag = false;
    }
    //start_flags[dev_id] = start_flag;   

    pcscGPU[dev_id].endCol = get_row_from_index(pcscNuma[numa_id].n, pcscNuma[numa_id].colPtr, pcscGPU[dev_id].endIdx);
    // Mark imcomplete rows
    // True: imcomplete
    if (pcscGPU[dev_id].endIdx < pcscNuma[numa_id].colPtr[pcscGPU[dev_id].endCol + 1] - 1)  {
      pcscGPU[dev_id].endFlag = true;
    } else {
      pcscGPU[dev_id].endFlag = false;
    }
    
    // Cacluclate dimensions
    pcscGPU[dev_id].m = m;
    pcscGPU[dev_id].n = pcscGPU[dev_id].endCol - pcscGPU[dev_id].startCol + 1;
    pcscGPU[dev_id].nnz = pcscGPU[dev_id].endIdx - pcscGPU[dev_id].startIdx + 1;

    pcscGPU[dev_id].val = &(pcscNuma[numa_id].val[pcscGPU[dev_id].startIdx]);
    pcscGPU[dev_id].colPtr = &(pcscNuma[numa_id].colPtr[pcscGPU[dev_id].startCol]);
    pcscGPU[dev_id].rowIdx = &(pcscNuma[numa_id].rowIdx[pcscGPU[dev_id].startIdx]);
    pcscGPU[dev_id].x = &(pcscNuma[numa_id].x[pcscGPU[dev_id].startCol]);
    pcscGPU[dev_id].y = pcscNuma[numa_id].y;

    part_time = get_time() - tmp_time;

    checkCudaErrors(hipMalloc((void**)&pcscGPU[dev_id].dval,    pcscGPU[dev_id].nnz     * sizeof(double)));
    checkCudaErrors(hipMalloc((void**)&pcscGPU[dev_id].dcolPtr, (pcscGPU[dev_id].n + 1) * sizeof(int)   ));
    checkCudaErrors(hipMalloc((void**)&pcscGPU[dev_id].drowIdx, pcscGPU[dev_id].nnz     * sizeof(int)   ));
    checkCudaErrors(hipMalloc((void**)&pcscGPU[dev_id].dx,      pcscGPU[dev_id].n       * sizeof(double))); 
    checkCudaErrors(hipMalloc((void**)&pcscGPU[dev_id].dy,      pcscGPU[dev_id].m       * sizeof(double)));

    checkCudaErrors(hipHostMalloc((void**)&(pcscGPU[dev_id].py), pcscGPU[dev_id].m * sizeof(double)));
  
    if (part_opt == 1) {
      checkCudaErrors(hipEventRecord(comm_start, stream));
      checkCudaErrors(hipMemcpyAsync(pcscGPU[dev_id].dcolPtr, pcscGPU[dev_id].colPtr, (pcscGPU[dev_id].n + 1) * sizeof(int), hipMemcpyHostToDevice, stream)); 
      checkCudaErrors(hipEventRecord(comm_stop, stream));
      checkCudaErrors(hipDeviceSynchronize());

      tmp_time = get_time();
      calcCscColPtr(pcscGPU[dev_id].dcolPtr, pcscGPU[dev_id].n, pcscGPU[dev_id].startIdx, pcscGPU[dev_id].nnz, stream);
      checkCudaErrors(hipDeviceSynchronize());
      part_time += get_time() - tmp_time; 
    }

    if (part_opt == 0) {
      checkCudaErrors(hipHostMalloc((void**)&(pcscGPU[dev_id].host_cscColPtr), (pcscGPU[dev_id].n + 1)*sizeof(int)));

      tmp_time = get_time();
      pcscGPU[dev_id].host_cscColPtr[0] = 0;
      pcscGPU[dev_id].host_cscColPtr[pcscGPU[dev_id].n] = pcscGPU[dev_id].nnz;
      for (int j = 1; j < pcscGPU[dev_id].n; j++) {
        pcscGPU[dev_id].host_cscColPtr[j] = pcscGPU[dev_id].colPtr[j] - pcscGPU[dev_id].startIdx;
      }
      part_time += get_time() - tmp_time;
      checkCudaErrors(hipEventRecord(comm_start, stream));
      checkCudaErrors(hipMemcpyAsync(pcscGPU[dev_id].dcolPtr, pcscGPU[dev_id].host_cscColPtr, (pcscGPU[dev_id].n + 1) * sizeof(int), hipMemcpyHostToDevice, stream));
      checkCudaErrors(hipEventRecord(comm_stop, stream));
      checkCudaErrors(hipHostFree(pcscGPU[dev_id].host_cscColPtr));
    }
    
    checkCudaErrors(hipEventSynchronize(comm_stop));
    elapsedTime = 0.0;
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, comm_start, comm_stop));
    elapsedTime /= 1000.0;
    comm_time += elapsedTime;


    // print_vec(pcscGPU[dev_id].val, pcscGPU[dev_id].nnz, "cscVal"+to_string(dev_id));
    // print_vec(pcscGPU[dev_id].colPtr, pcscGPU[dev_id].n + 1, "colPtr"+to_string(dev_id));
    // print_vec(pcscGPU[dev_id].rowIdx, pcscGPU[dev_id].nnz, "rowIdx"+to_string(dev_id));
    // print_vec(pcscGPU[dev_id].x, pcscGPU[dev_id].n, "x"+to_string(dev_id));
    // print_vec(pcscGPU[dev_id].y, pcscGPU[dev_id].m, "y_before"+to_string(dev_id));
    // printf("dev_id %d, alpha %f, beta %f\n", dev_id, *alpha, *beta);
    // host_cscColPtr[0] = 0;
    // host_cscColPtr[dev_n] = dev_nnz;
    // for (int j = 1; j < dev_m; j++) {
    //   host_cscColPtr[j] = (int)(cscColPtr[start_col + j] - start_idx);
    // }
    //host_cscRowIndex = cscRowIndex[start_idx];
    // host_x = &x[start_col];
    // host_y = y;
  


  
    // preparing GPU env
    
    
    


    // double * dev_csrVal;
    // int * dev_csrRowPtr;
    // int * dev_csrColIndex;
    // checkCudaErrors(hipMalloc((void**)&dev_csrVal,    pcscGPU[dev_id].nnz     * sizeof(double)));
    // checkCudaErrors(hipMalloc((void**)&dev_csrRowPtr, (pcscGPU[dev_id].m + 1) * sizeof(int)   ));
    // checkCudaErrors(hipMalloc((void**)&dev_csrColIndex, pcscGPU[dev_id].nnz     * sizeof(int) ));

    // double * A;
    // int lda = m;
    // checkCudaErrors(hipMalloc((void**)&A, m * n * sizeof(double)));


     
    // printf("dev_id %d, part_time = %f\n", dev_id, part_time); 


    // #pragma omp barrier
    // tmp_time = get_time();
    checkCudaErrors(hipEventRecord(comm_start, stream));
    hipMemcpyAsync(pcscGPU[dev_id].dval,    pcscGPU[dev_id].val,    pcscGPU[dev_id].nnz * sizeof(double), hipMemcpyHostToDevice, stream);
    // hipMemcpyAsync(pcscGPU[dev_id].dcolPtr, pcscGPU[dev_id].colPtr, (pcscGPU[dev_id].n + 1) * sizeof(int), hipMemcpyHostToDevice, stream); 
    hipMemcpyAsync(pcscGPU[dev_id].drowIdx, pcscGPU[dev_id].rowIdx, pcscGPU[dev_id].nnz * sizeof(int), hipMemcpyHostToDevice, stream); 
    hipMemcpyAsync(pcscGPU[dev_id].dx,      pcscGPU[dev_id].x,      pcscGPU[dev_id].n * sizeof(double),  hipMemcpyHostToDevice, stream); 
    hipMemcpyAsync(pcscGPU[dev_id].dy,      pcscGPU[dev_id].y,      pcscGPU[dev_id].m * sizeof(double), hipMemcpyHostToDevice, stream); 
    checkCudaErrors(hipEventRecord(comm_stop, stream));
    // checkCudaErrors(hipDeviceSynchronize());
    // print_vec_gpu(pcscGPU[dev_id].dval, pcscGPU[dev_id].nnz, "cscVal"+to_string(dev_id));
    // print_vec_gpu(pcscGPU[dev_id].dcolPtr, pcscGPU[dev_id].n + 1, "colPtr"+to_string(dev_id));
    // print_vec_gpu(pcscGPU[dev_id].drowIdx, pcscGPU[dev_id].nnz, "rowIdx"+to_string(dev_id));
    // print_vec_gpu(pcscGPU[dev_id].dx, pcscGPU[dev_id].n, "x"+to_string(dev_id));
    // print_vec_gpu(pcscGPU[dev_id].dy, pcscGPU[dev_id].m, "y_before"+to_string(dev_id));
    // printf("dev_id %d, alpha %f, beta %f\n", dev_id, *alpha, *beta);

    
  //   time_comm = get_time() - curr_time;
  //   curr_time = get_time();

  //   err = 0;
    // if (kernel == 1) {

    

    // csc2csrGPU(handle, m, n, nnz, A, lda,
    //              pcscGPU[dev_id].dval, pcscGPU[dev_id].dcolPtr, pcscGPU[dev_id].drowIdx,
    //              dev_csrVal, dev_csrRowPtr, dev_csrColIndex); 


    // print_vec_gpu(dev_csrVal, pcscGPU[dev_id].nnz, "csrVal"+to_string(dev_id));
    // print_vec_gpu(dev_csrRowPtr, pcscGPU[dev_id].m + 1, "csrRowPtr"+to_string(dev_id));
    // print_vec_gpu(dev_csrColIndex, pcscGPU[dev_id].nnz, "csrColIndex"+to_string(dev_id));
    // print_vec_gpu(pcscGPU[dev_id].dx, pcscGPU[dev_id].n, "x"+to_string(dev_id));
    // print_vec_gpu(pcscGPU[dev_id].dy, pcscGPU[dev_id].m, "y_before"+to_string(dev_id));
    // printf("dev_id %d, alpha %f, beta %f\n", dev_id, *alpha, *beta);

    checkCudaErrors(hipEventRecord(comp_start, stream));
    checkCudaErrors(hipsparseDcsrmv(handle,HIPSPARSE_OPERATION_TRANSPOSE, 
                            pcscGPU[dev_id].n, pcscGPU[dev_id].m, pcscGPU[dev_id].nnz, 
                            alpha, descr, 
                            pcscGPU[dev_id].dval, pcscGPU[dev_id].dcolPtr, pcscGPU[dev_id].drowIdx,
                            pcscGPU[dev_id].dx, beta, pcscGPU[dev_id].dy));
    checkCudaErrors(hipEventRecord(comp_stop, stream));

    // checkCudaErrors(hipsparseDcsrmv(handle,HIPSPARSE_OPERATION_NON_TRANSPOSE, 
    //                         pcscGPU[dev_id].m, pcscGPU[dev_id].n, pcscGPU[dev_id].nnz, 
    //                         alpha, descr, dev_csrVal, 
    //                         dev_csrRowPtr, dev_csrColIndex, 
    //                         pcscGPU[dev_id].dx, beta, pcscGPU[dev_id].dy));
      
    checkCudaErrors(hipEventSynchronize(comm_stop));
    elapsedTime = 0.0;
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, comm_start, comm_stop));
    elapsedTime /= 1000.0;
    comm_time += elapsedTime;

    checkCudaErrors(hipEventSynchronize(comp_stop));
    elapsedTime = 0.0;
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, comp_start, comp_stop));
    elapsedTime /= 1000.0;
    comp_time += elapsedTime;
  
    checkCudaErrors(hipDeviceSynchronize());
    // print_vec_gpu(pcscGPU[dev_id].dy, pcscGPU[dev_id].m, "y_after"+to_string(dev_id));
    // printf("omp thread %d, time %f\n", dev_id, get_time() - tmp_time);
    //comp_time = get_time() - tmp_time;


    tmp_time = get_time();

    if (merg_opt == 0 || merg_opt == 1) {
      checkCudaErrors(hipMemcpyAsync(pcscGPU[dev_id].py, pcscGPU[dev_id].dy, 
                      pcscGPU[dev_id].m * sizeof(double), hipMemcpyDeviceToHost, stream)); 

      checkCudaErrors(hipDeviceSynchronize());
      #pragma omp barrier
      //
      // print_vec(pcscGPU[dev_id].py, m, "py"+to_string(dev_id));
      if (dev_id == 0) {
        for (int d = 0; d < ngpu; d++) {
          for (int i = 0; i < m; i++) {
            y[i] += pcscGPU[d].py[i];
          }
          // print_vec(pcscGPU[d].py, m, "py-after"+to_string(d));
          // print_vec(y, m, "y"+to_string(d));
        }
      }
    }

    if (merg_opt == 1) {
      // to be done
    }
    merg_time = get_time() - tmp_time;

    checkCudaErrors(hipFree(pcscGPU[dev_id].dval));
    checkCudaErrors(hipFree(pcscGPU[dev_id].dcolPtr));
    checkCudaErrors(hipFree(pcscGPU[dev_id].drowIdx));
    checkCudaErrors(hipFree(pcscGPU[dev_id].dx));
    checkCudaErrors(hipFree(pcscGPU[dev_id].dy));

    checkCudaErrors(hipHostFree(pcscGPU[dev_id].py));
    

    checkCudaErrors(hipEventDestroy(comp_start));
    checkCudaErrors(hipEventDestroy(comp_stop));
    checkCudaErrors(hipEventDestroy(comm_start));
    checkCudaErrors(hipEventDestroy(comm_stop));
          
    checkCudaErrors(hipsparseDestroyMatDescr(descr));
    checkCudaErrors(hipsparseDestroy(handle));
    checkCudaErrors(hipStreamDestroy(stream));

  }

  for (int numa_id = 0; numa_id < numaContext.numNumaNodes; numa_id++) {
    checkCudaErrors(hipHostFree(pcscNuma[numa_id].val));
    checkCudaErrors(hipHostFree(pcscNuma[numa_id].colPtr));
    checkCudaErrors(hipHostFree(pcscNuma[numa_id].rowIdx));
    checkCudaErrors(hipHostFree(pcscNuma[numa_id].x));
    checkCudaErrors(hipHostFree(pcscNuma[numa_id].y));
  }

  // print_vec(y, m, "y_all");

  spmv_ret ret;
  ret.numa_part_time = numa_part_time;
  ret.comp_time = comp_time;
  ret.comm_time = 0.0;
  ret.part_time = part_time;
  ret.merg_time = merg_time;
  return ret;
}

