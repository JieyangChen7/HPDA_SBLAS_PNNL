#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <ctime>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "hipsparse.h"
#include <iostream>
#include <cmath>
#include <float.h>
//#include "anonymouslib_cuda.h"
#include "spmv_kernel.h"
using namespace std;

spmv_ret spMV_mgpu_baseline(int m, int n, int nnz, double * alpha,
            double * csrVal, int * csrRowPtr, int * csrColIndex, 
            double * x, double * beta,
            double * y,
            int ngpu){

  double curr_time = 0.0;
  double numa_part_time = 0.0;
  double part_time = 0.0;
  double comp_time = 0.0;
  double comm_time = 0.0;
  double merg_time = 0.0;

  hipEvent_t * comp_start = new hipEvent_t[ngpu];
  hipEvent_t * comp_stop = new hipEvent_t[ngpu];
  
  hipEvent_t * comm_start = new hipEvent_t[ngpu];
  hipEvent_t * comm_stop = new hipEvent_t[ngpu];

  curr_time = get_time();

  hipStream_t * stream = new hipStream_t [ngpu];

  hipError_t * cudaStat1 = new hipError_t[ngpu];
  hipError_t * cudaStat2 = new hipError_t[ngpu];
  hipError_t * cudaStat3 = new hipError_t[ngpu];
  hipError_t * cudaStat4 = new hipError_t[ngpu];
  hipError_t * cudaStat5 = new hipError_t[ngpu];
  hipError_t * cudaStat6 = new hipError_t[ngpu];

  hipsparseStatus_t * status = new hipsparseStatus_t[ngpu];
  hipsparseHandle_t * handle = new hipsparseHandle_t[ngpu];
  hipsparseMatDescr_t * descr = new hipsparseMatDescr_t[ngpu];

  int  * start_row  = new int[ngpu];
  int  * end_row    = new int[ngpu];
    
  int * dev_m            = new int      [ngpu];
  int * dev_n            = new int      [ngpu];
  int * dev_nnz          = new int      [ngpu];
  int ** host_csrRowPtr  = new int    * [ngpu];
  int ** dev_csrRowPtr   = new int    * [ngpu];
  int ** dev_csrColIndex = new int    * [ngpu];
  double ** dev_csrVal   = new double * [ngpu];


  double ** dev_x = new double * [ngpu];
  double ** dev_y = new double * [ngpu];

  curr_time = get_time();
  for (int d = 0; d < ngpu; d++){
    start_row[d] = floor((d)     * m / ngpu);
    end_row[d]   = floor((d + 1) * m / ngpu) - 1;
    dev_m[d]   = end_row[d] - start_row[d] + 1;
    dev_n[d]   = n;
    dev_nnz[d] = (int)(csrRowPtr[end_row[d] + 1] - csrRowPtr[start_row[d]]);
  }
  part_time += get_time() - curr_time;
  
  for (int d = 0; d < ngpu; d++) {
    checkCudaErrors(hipHostMalloc((void**)& host_csrRowPtr[d], (dev_m[d]+1) * sizeof(int)));
  }

  curr_time = get_time();

  for (int d = 0; d < ngpu; d++){
    for (int i = 0; i < dev_m[d] + 1; i++) {
      host_csrRowPtr[d][i] = (int)(csrRowPtr[start_row[d] + i] - csrRowPtr[start_row[d]]);
    }

  }


  part_time += get_time() - curr_time;

  for (int d = 0; d < ngpu; d++){
    checkCudaErrors(hipSetDevice(d));
    checkCudaErrors(hipStreamCreate(&(stream[d])));
    checkCudaErrors(hipsparseCreate(&(handle[d]))); 
    checkCudaErrors(hipsparseSetStream(handle[d], stream[d]));
    checkCudaErrors(hipsparseCreateMatDescr(&descr[d]));
    checkCudaErrors(hipsparseSetMatType(descr[d],HIPSPARSE_MATRIX_TYPE_GENERAL)); 
    checkCudaErrors(hipsparseSetMatIndexBase(descr[d],HIPSPARSE_INDEX_BASE_ZERO)); 

    checkCudaErrors(hipEventCreate(&(comp_start[d])));
    checkCudaErrors(hipEventCreate(&(comp_stop[d])));
    checkCudaErrors(hipEventCreate(&(comm_start[d])));
    checkCudaErrors(hipEventCreate(&(comm_stop[d])));

    checkCudaErrors(hipMalloc((void**)&dev_csrRowPtr[d],   (dev_m[d] + 1) * sizeof(int)));
    checkCudaErrors(hipMalloc((void**)&dev_csrColIndex[d], dev_nnz[d] * sizeof(int))); 
    checkCudaErrors(hipMalloc((void**)&dev_csrVal[d],      dev_nnz[d] * sizeof(double))); 
    checkCudaErrors(hipMalloc((void**)&dev_x[d],           dev_n[d] * sizeof(double))); 
    checkCudaErrors(hipMalloc((void**)&dev_y[d],           dev_m[d] * sizeof(double))); 
  }

  //curr_time = get_time();
  
  for (int d = 0; d < ngpu; d++){
    checkCudaErrors(hipSetDevice(d));
    hipEventRecord(comm_start[d], stream[d]);
    checkCudaErrors(hipMemcpyAsync(dev_csrRowPtr[d],   host_csrRowPtr[d],                  (size_t)((dev_m[d] + 1) * sizeof(int)), hipMemcpyHostToDevice, stream[d]));
    checkCudaErrors(hipMemcpyAsync(dev_csrColIndex[d], &csrColIndex[csrRowPtr[start_row[d]]], (size_t)(dev_nnz[d] * sizeof(int)),   hipMemcpyHostToDevice, stream[d])); 
    checkCudaErrors(hipMemcpyAsync(dev_csrVal[d],      &csrVal[csrRowPtr[start_row[d]]],      (size_t)(dev_nnz[d] * sizeof(double)), hipMemcpyHostToDevice, stream[d]));
    checkCudaErrors(hipMemcpyAsync(dev_y[d], &y[start_row[d]], (size_t)(dev_m[d]*sizeof(double)), hipMemcpyHostToDevice, stream[d])); 
    checkCudaErrors(hipMemcpyAsync(dev_x[d], x,                (size_t)(dev_n[d]*sizeof(double)), hipMemcpyHostToDevice, stream[d])); 
    hipEventRecord(comm_stop[d], stream[d]);
  }
  for (int d = 0; d < ngpu; ++d) {
    checkCudaErrors(hipSetDevice(d));
    hipEventRecord(comp_start[d], stream[d]);
    checkCudaErrors(hipsparseDcsrmv(handle[d],HIPSPARSE_OPERATION_NON_TRANSPOSE, 
                   dev_m[d], dev_n[d], dev_nnz[d], 
                   alpha, descr[d], dev_csrVal[d], 
                   dev_csrRowPtr[d], dev_csrColIndex[d], 
                   dev_x[d], beta, dev_y[d]));
    hipEventRecord(comp_stop[d], stream[d]);     
  }

  for (int d = 0; d < ngpu; ++d) {
    checkCudaErrors(hipSetDevice(d));
    hipEventSynchronize(comm_stop[d]);
    float elapsedTime = 0.0;
    hipEventElapsedTime(&elapsedTime, comm_start[d], comm_stop[d]);
    elapsedTime /= 1000.0;
    if (elapsedTime > comm_time) comm_time = elapsedTime;

    

    hipEventSynchronize(comp_stop[d]);
    elapsedTime = 0.0;
    hipEventElapsedTime(&elapsedTime, comp_start[d], comp_stop[d]);
    elapsedTime /= 1000.0;
    if (elapsedTime > comp_time) comp_time = elapsedTime;

    printf("dev %d, elapsedTime1 %f comp_time %f\n", d, elapsedTime, comp_time);

    checkCudaErrors(hipDeviceSynchronize());
  }
  //comp_time = get_time() - curr_time;

  curr_time = get_time();
  for (int d = 0; d < ngpu; d++) {
    checkCudaErrors(hipSetDevice(d));
    checkCudaErrors(hipMemcpyAsync( &y[start_row[d]], dev_y[d], (size_t)(dev_m[d]*sizeof(double)),  hipMemcpyDeviceToHost, stream[d]));
  }
  for (int d = 0; d < ngpu; d++) {
    checkCudaErrors(hipSetDevice(d));
    checkCudaErrors(hipDeviceSynchronize());
  }
  merg_time = get_time() - curr_time;

  for (int d = 0; d < ngpu; d++) {
    hipSetDevice(d);
    hipFree(dev_csrVal[d]);
    hipFree(dev_csrRowPtr[d]);
    hipFree(dev_csrColIndex[d]);
    hipFree(dev_x[d]);
    hipFree(dev_y[d]);
    hipEventDestroy(comp_start[d]);
    hipEventDestroy(comp_stop[d]);
    hipEventDestroy(comm_start[d]);
    hipEventDestroy(comm_stop[d]);
  }

  delete[] dev_csrVal;
  delete[] dev_csrRowPtr;
  delete[] dev_csrColIndex;
  delete[] dev_x;
  delete[] dev_y;
  delete[] host_csrRowPtr;
  delete[] start_row;
  delete[] end_row;
  delete[] comp_start;
  delete[] comp_stop;
  delete[] comm_start;
  delete[] comm_stop;
  
  spmv_ret ret;
  ret.numa_part_time = numa_part_time;
  ret.part_time = part_time;
  ret.comp_time = comp_time;
  ret.comm_time = comm_time;
  ret.merg_time = merg_time;
  return ret;

}
