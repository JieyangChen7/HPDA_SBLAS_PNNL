#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <ctime>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "hipsparse.h"
#include <iostream>
#include <cmath>
#include <float.h>
#include "common_cuda.h"
//#include "anonymouslib_cuda.h"
#include "spmv_kernel.h"
#include <limits>
#include <sstream>
#include <string>
using namespace std;

int get_row_from_index(int n, int * a, int idx) {
  int l = 0;
  int r = n;
  int m = l + (r - l) / 2;
  while (l < r - 1) {
    //cout << "l = " << l <<endl;
    //cout << "r = " << r <<endl;
    int m = l + (r - l) / 2;
    //cout << "m = " << m <<endl;
    if (idx < a[m]) {
      r = m;
    } else if (idx >= a[m]) {
      l = m;
    } 
    //else {
    //  cout << "1st return: " << m << endl;
    //  return m;
    //}
  }
  //cout << "a[" << l << "] = " <<  a[l] << endl;
  //cout << " a[" << r << "] = " << a[r] << endl;
  //cout << " idx = " << idx << endl;
  if (idx == a[l]) return l;
  if (idx == a[r]) return r;
  return l;

}

double get_time()
{
  struct timeval tp;
  gettimeofday(&tp, NULL);
  double ms = (double)tp.tv_sec * 1000 + (double)tp.tv_usec / 1000; //get current timestamp in milliseconds
  //return 0.00001;
  return ms / 1000;
}


double get_gpu_availble_mem(int ngpu) {
  size_t uCurAvailMemoryInBytes;
  size_t uTotalMemoryInBytes;
  

  double min_mem = numeric_limits<double>::max();
  int device;
  for (device = 0; device < ngpu; ++device) 
  {
    hipSetDevice(device);
    hipMemGetInfo(&uCurAvailMemoryInBytes, &uTotalMemoryInBytes);
    cout << uCurAvailMemoryInBytes << "/" << uTotalMemoryInBytes << endl;
    double aval_mem = (double)uCurAvailMemoryInBytes/1e9;
    cout << aval_mem << endl;
    if (aval_mem < min_mem) {
      min_mem = aval_mem;
    }
      // hipDeviceProp_t deviceProp;
      // hipGetDeviceProperties(&deviceProp, device);
      // printf("Device %d has compute capability %d.%d.\n",
      //        device, deviceProp.major, deviceProp.minor);
  }


  return min_mem;
}

void print_vec(double * a, int n, string s) {
  ostringstream ss;
  ss << s << ": ";
  for (int i = 0; i < n ; i++) {
    ss << a[i] << " ";
  }
  ss << endl;
  string res = ss.str();
  cout << res;
}

void print_vec(int * a, int n, string s) {
  ostringstream ss;
  ss << s << ": ";
  for (int i = 0; i < n ; i++) {
    ss << a[i] << " ";
  }
  ss << endl;
  string res = ss.str();
  cout << res;
}



void print_vec_gpu(double * a, int n, string s) {
  double * ha = new double[n]; 
  hipMemcpy(ha, a, n*sizeof(double), hipMemcpyDeviceToHost);
  print_vec(ha, n, s);
  delete [] ha;
}

void print_vec_gpu(int * a, int n, string s) {
  int * ha = new int[n];
  hipMemcpy(ha, a, n*sizeof(int), hipMemcpyDeviceToHost);
  print_vec(ha, n, s);
  delete [] ha;
}


void csr2csrNcsc(int m, int n, int nnz,
             double * cooVal, int * cooRowIdx, int * cooColIdx,
             double * csrVal, int * csrRowPtr, int * csrColIdx,
             double * cscVal, int * cscColPtr, int * cscRowIdx) {

  printf("converting: %d, %d\n", m, n);

  double * A = new double[m * n];
  for (int i = 0; i < m; i++) {
    for (int j = 0; j < n; j++) {
      A[i * n + j] = 0.0;
    }
  }

  for (int i = 0; i < nnz; i++) {
    A[cooRowIdx[i] * n + cooColIdx[i]] = cooVal[i];
  }

  for (int i = 0; i < m; i++) {
    print_vec(&(A[i * n]), n, "A");
  }

  int p = 0;
  csrRowPtr[0] = 0;
  for (int i = 0; i < m; i++) {
    for (int j = 0; j < n; j++) {
      if (A[i * n + j] != 0) {
        csrVal[p] = A[i * n + j];
        csrColIdx[p] = j;
        //printf("add %f, %d\n", csrVal[p], csrColIdx[p]);
        p++;
      }
    }
    //printf("add to row %d\n", i+1);
    csrRowPtr[i + 1] = p;
    //printf("row %d\n", p);
  }

  

  p = 0;
  cscColPtr[0] = 0;
  for (int j = 0; j < n; j++) {
    for (int i = 0; i < m; i++) {
      if (A[i * n + j] != 0) {
        cscVal[p] = A[i * n + j];
        cscRowIdx[p] = i;
        p++;
      }
    }
    cscColPtr[j + 1] = p;
  }

  printf("done converting\n");
  delete [] A;

  // double * dcsrVal;
  // int * dcsrRowPtr;
  // int * dcsrColIdx;
  // double * dcscVal;
  // int * dcscColPtr;
  // int * dcscRowIdx;


  // checkCudaErrors(hipMalloc((void**)&dcsrVal, nnz * sizeof(double)));
  // checkCudaErrors(hipMalloc((void**)&dcsrRowPtr, (m+1) * sizeof(int)));
  // checkCudaErrors(hipMalloc((void**)&dcsrColIdx, nnz * sizeof(int)));

  // checkCudaErrors(hipMalloc((void**)&dcscVal, nnz * sizeof(double)));
  // checkCudaErrors(hipMalloc((void**)&dcscColPtr, (n+1) * sizeof(int)));
  // checkCudaErrors(hipMalloc((void**)&dcscRowIdx, nnz * sizeof(int)));

  // checkCudaErrors(hipMemcpy(dcsrVal, csrVal, nnz * sizeof(double),
  //                             hipMemcpyHostToDevice));
  // checkCudaErrors(hipMemcpy(dcsrRowPtr, csrRowPtr, (m+1) * sizeof(int),
  //                             hipMemcpyHostToDevice));
  // checkCudaErrors(hipMemcpy(dcsrColIdx, csrColIdx, nnz * sizeof(int),
  //                             hipMemcpyHostToDevice));
  // checkCudaErrors(hipMemcpy(dcscVal, cscVal, nnz * sizeof(double),
  //                             hipMemcpyHostToDevice));
  // checkCudaErrors(hipMemcpy(dcscColPtr, cscColPtr, (n+1) * sizeof(int),
  //                             hipMemcpyHostToDevice));
  // checkCudaErrors(hipMemcpy(dcscRowIdx, cscRowIdx, nnz * sizeof(int),
  //                             hipMemcpyHostToDevice));

  // csr2csc_gpu(m, n, nnz,
  //             dcsrVal, dcsrRowPtr, dcsrColIdx,
  //             dcscVal, dcscColPtr, dcscRowIdx);

  
  // checkCudaErrors(hipMemcpy(csrVal, dcsrVal, nnz * sizeof(double),
  //                             hipMemcpyDeviceToHost));
  // checkCudaErrors(hipMemcpy(csrRowPtr, dcsrRowPtr, (m+1) * sizeof(int),
  //                             hipMemcpyDeviceToHost));
  // checkCudaErrors(hipMemcpy(csrColIdx, dcsrColIdx, nnz * sizeof(int),
  //                             hipMemcpyDeviceToHost));
  // checkCudaErrors(hipMemcpy(dcscVal, cscVal, nnz * sizeof(double),
  //                             hipMemcpyDeviceToHost));
  // checkCudaErrors(hipMemcpy(dcscColPtr, cscColPtr, (n+1) * sizeof(int),
  //                             hipMemcpyDeviceToHost));
  // checkCudaErrors(hipMemcpy(dcscRowIdx, cscRowIdx, nnz * sizeof(int),
  //                             hipMemcpyDeviceToHost));

  // checkCudaErrors(hipFree(dcsrVal));
  // checkCudaErrors(hipFree(dcsrRowPtr));
  // checkCudaErrors(hipFree(dcsrColIdx));
  // checkCudaErrors(hipFree(dcscVal));
  // checkCudaErrors(hipFree(dcscColPtr));
  // checkCudaErrors(hipFree(dcscRowIdx));

}


// void csr2csc(int m, int n, int nnz,
//              double * csrVal, int * csrRowPtr, int * csrColIdx,
//              double * cscVal, int * cscColPtr, int * cscRowIdx) {

  



//   double * dcsrVal;
//   int * dcsrRowPtr;
//   int * dcsrColIdx;
//   double * dcscVal;
//   int * dcscColPtr;
//   int * dcscRowIdx;


//   checkCudaErrors(hipMalloc((void**)&dcsrVal, nnz * sizeof(double)));
//   checkCudaErrors(hipMalloc((void**)&dcsrRowPtr, (m+1) * sizeof(int)));
//   checkCudaErrors(hipMalloc((void**)&dcsrColIdx, nnz * sizeof(int)));

//   checkCudaErrors(hipMalloc((void**)&dcscVal, nnz * sizeof(double)));
//   checkCudaErrors(hipMalloc((void**)&dcscColPtr, (n+1) * sizeof(int)));
//   checkCudaErrors(hipMalloc((void**)&dcscRowIdx, nnz * sizeof(int)));

//   checkCudaErrors(hipMemcpy(dcsrVal, csrVal, nnz * sizeof(double),
//                               hipMemcpyHostToDevice));
//   checkCudaErrors(hipMemcpy(dcsrRowPtr, csrRowPtr, (m+1) * sizeof(int),
//                               hipMemcpyHostToDevice));
//   checkCudaErrors(hipMemcpy(dcsrColIdx, csrColIdx, nnz * sizeof(int),
//                               hipMemcpyHostToDevice));
//   checkCudaErrors(hipMemcpy(dcscVal, cscVal, nnz * sizeof(double),
//                               hipMemcpyHostToDevice));
//   checkCudaErrors(hipMemcpy(dcscColPtr, cscColPtr, (n+1) * sizeof(int),
//                               hipMemcpyHostToDevice));
//   checkCudaErrors(hipMemcpy(dcscRowIdx, cscRowIdx, nnz * sizeof(int),
//                               hipMemcpyHostToDevice));

//   csr2csc_gpu(m, n, nnz,
//               dcsrVal, dcsrRowPtr, dcsrColIdx,
//               dcscVal, dcscColPtr, dcscRowIdx);

  
//   checkCudaErrors(hipMemcpy(csrVal, dcsrVal, nnz * sizeof(double),
//                               hipMemcpyDeviceToHost));
//   checkCudaErrors(hipMemcpy(csrRowPtr, dcsrRowPtr, (m+1) * sizeof(int),
//                               hipMemcpyDeviceToHost));
//   checkCudaErrors(hipMemcpy(csrColIdx, dcsrColIdx, nnz * sizeof(int),
//                               hipMemcpyDeviceToHost));
//   checkCudaErrors(hipMemcpy(dcscVal, cscVal, nnz * sizeof(double),
//                               hipMemcpyDeviceToHost));
//   checkCudaErrors(hipMemcpy(dcscColPtr, cscColPtr, (n+1) * sizeof(int),
//                               hipMemcpyDeviceToHost));
//   checkCudaErrors(hipMemcpy(dcscRowIdx, cscRowIdx, nnz * sizeof(int),
//                               hipMemcpyDeviceToHost));

//   checkCudaErrors(hipFree(dcsrVal));
//   checkCudaErrors(hipFree(dcsrRowPtr));
//   checkCudaErrors(hipFree(dcsrColIdx));
//   checkCudaErrors(hipFree(dcscVal));
//   checkCudaErrors(hipFree(dcscColPtr));
//   checkCudaErrors(hipFree(dcscRowIdx));

// }



void csr2csc_gpu(int m, int n, int nnz,
                 double * csrVal, int * csrRowPtr, int * csrColIdx,
                 double * cscVal, int * cscColPtr, int * cscRowIdx) {
  hipStream_t stream;
  hipsparseStatus_t status;
  hipsparseHandle_t handle;
  hipsparseMatDescr_t descr;
  checkCudaErrors(hipStreamCreate(&stream));
  checkCudaErrors(hipsparseCreate(&handle)); 
  checkCudaErrors(hipsparseSetStream(handle, stream));
  checkCudaErrors(hipsparseCreateMatDescr(&descr));
  checkCudaErrors(hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL)); 
  checkCudaErrors(hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO));

  double * A;
  int lda = m;
  checkCudaErrors(hipMalloc((void**)&A, lda * n * sizeof(double)));

  int * nnzPerCol = new int[n];
  int * nnzTotalDevHostPtr = new int[1];
  checkCudaErrors(hipsparseDcsr2dense(handle, m, n, descr,
                                     csrVal, csrRowPtr, csrColIdx,
                                     A, lda));

  checkCudaErrors(hipsparseDnnz(handle, HIPSPARSE_DIRECTION_COLUMN,
                               m, n, descr, A, lda, nnzPerCol, nnzTotalDevHostPtr));

  checkCudaErrors(hipsparseDdense2csc(handle, m, n, descr, 
                                     A, lda, nnzPerCol,
                                     cscVal, cscColPtr, cscRowIdx));

  // checkCudaErrors(hipsparseDcsr2csc(handle, m, n, nnz,
  //                                   csrVal, csrRowPtr, csrColIdx,
  //                                   cscVal, cscColPtr, cscRowIdx,
  //                                   HIPSPARSE_ACTION_NUMERIC,
  //                                   HIPSPARSE_INDEX_BASE_ZERO));

  checkCudaErrors(hipDeviceSynchronize());
  // checkCudaErrors(hipFree(buffer));
  // delete [] P;

  checkCudaErrors(hipsparseDestroyMatDescr(descr));
  checkCudaErrors(hipsparseDestroy(handle));
  checkCudaErrors(hipStreamDestroy(stream));

  
}


void csc2csr_gpu(hipsparseHandle_t handle, int m, int n, int nnz, double * A, int lda, 
                 double * cscVal, int * cscColPtr, int * cscRowIdx,
                 double * csrVal, int * csrRowPtr, int * csrColIdx) {
  // hipStream_t stream;
  // hipsparseStatus_t status;
  // hipsparseHandle_t handle;
  hipsparseMatDescr_t descr;
  // checkCudaErrors(hipStreamCreate(&stream));
  // checkCudaErrors(hipsparseCreate(&handle)); 
  // checkCudaErrors(hipsparseSetStream(handle, stream));
  checkCudaErrors(hipsparseCreateMatDescr(&descr));
  checkCudaErrors(hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL)); 
  checkCudaErrors(hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO));

  // double * A;
  // int lda = m;
  // checkCudaErrors(hipMalloc((void**)&A, lda * n * sizeof(double)));

  int * nnzPerRow = new int[m];
  int * nnzTotalDevHostPtr = new int[1];
  checkCudaErrors(hipsparseDcsc2dense(handle, m, n, descr,
                                     cscVal, cscRowIdx, cscColPtr,
                                     A, lda));

  checkCudaErrors(hipsparseDnnz(handle, HIPSPARSE_DIRECTION_ROW,
                               m, n, descr, A, lda, nnzPerRow, nnzTotalDevHostPtr));

  checkCudaErrors(hipsparseDdense2csr(handle, m, n, descr, 
                                     A, lda, nnzPerRow,
                                     csrVal, csrRowPtr, csrColIdx));

  // checkCudaErrors(hipsparseDcsr2csc(handle, m, n, nnz,
  //                                   csrVal, csrRowPtr, csrColIdx,
  //                                   cscVal, cscColPtr, cscRowIdx,
  //                                   HIPSPARSE_ACTION_NUMERIC,
  //                                   HIPSPARSE_INDEX_BASE_ZERO));

  // checkCudaErrors(hipDeviceSynchronize());
  // checkCudaErrors(hipFree(buffer));
  // delete [] P;

  // checkCudaErrors(hipsparseDestroyMatDescr(descr));
  // checkCudaErrors(hipsparseDestroy(handle));
  // checkCudaErrors(hipStreamDestroy(stream));

  
}

void sortCOORow(int m, int n, int nnz,
                double * cooVal, int * cooRowIdx, int * cooColIdx) {
  hipStream_t stream;
  hipsparseStatus_t status;
  hipsparseHandle_t handle;

  checkCudaErrors(hipStreamCreate(&stream));
  checkCudaErrors(hipsparseCreate(&handle)); 
  checkCudaErrors(hipsparseSetStream(handle, stream));

  double * dcooVal;
  int * dcooRowIdx;
  int * dcooColIdx;
  double * dcooValSorted;
  void * buffer;
  size_t bufferSize = 0;
  int * dP;

  checkCudaErrors(hipsparseXcoosort_bufferSizeExt(handle, m, n, nnz, 
                                                  dcooRowIdx, dcooColIdx,
                                                  &bufferSize));

  checkCudaErrors(hipMalloc((void**)&dcooVal, nnz * sizeof(double)));
  checkCudaErrors(hipMalloc((void**)&dcooRowIdx, nnz * sizeof(int)));
  checkCudaErrors(hipMalloc((void**)&dcooColIdx, nnz * sizeof(int)));
  checkCudaErrors(hipMalloc((void**)&dcooValSorted, nnz * sizeof(double)));
  checkCudaErrors(hipMalloc((void**)&buffer, bufferSize ));
  checkCudaErrors(hipMalloc((void**)&dP, nnz * sizeof(int)));

  checkCudaErrors(hipMemcpy(dcooVal, cooVal, nnz * sizeof(double), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(dcooRowIdx, cooRowIdx, nnz * sizeof(int), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(dcooColIdx, cooColIdx, nnz * sizeof(int), hipMemcpyHostToDevice));

  checkCudaErrors(hipsparseCreateIdentityPermutation(handle, nnz, dP));
  
  checkCudaErrors(hipsparseXcoosortByRow(handle, m, n, nnz, 
                                        dcooRowIdx, dcooColIdx,
                                        dP, buffer));
  checkCudaErrors(hipsparseDgthr(handle, nnz, dcooVal, dcooValSorted, dP,
                                HIPSPARSE_INDEX_BASE_ZERO));

  checkCudaErrors(hipMemcpy(cooVal, dcooValSorted, nnz * sizeof(double), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(cooRowIdx, dcooRowIdx, nnz * sizeof(int), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(cooColIdx, dcooColIdx, nnz * sizeof(int), hipMemcpyDeviceToHost));

  checkCudaErrors(hipFree(dcooVal));
  checkCudaErrors(hipFree(dcooRowIdx));
  checkCudaErrors(hipFree(dcooColIdx));
  checkCudaErrors(hipFree(dcooValSorted));
  checkCudaErrors(hipFree(buffer));
  checkCudaErrors(hipFree(dP));

}

void sortCOOCol(int m, int n, int nnz,
                double * cooVal, int * cooRowIdx, int * cooColIdx) {
  hipStream_t stream;
  hipsparseStatus_t status;
  hipsparseHandle_t handle;

  checkCudaErrors(hipStreamCreate(&stream));
  checkCudaErrors(hipsparseCreate(&handle)); 
  checkCudaErrors(hipsparseSetStream(handle, stream));

  double * dcooVal;
  int * dcooRowIdx;
  int * dcooColIdx;
  double * dcooValSorted;
  void * buffer;
  size_t bufferSize = 0;
  int * dP;

  checkCudaErrors(hipsparseXcoosort_bufferSizeExt(handle, m, n, nnz, 
                                                  dcooRowIdx, dcooColIdx,
                                                  &bufferSize));

  checkCudaErrors(hipMalloc((void**)&dcooVal, nnz * sizeof(double)));
  checkCudaErrors(hipMalloc((void**)&dcooRowIdx, nnz * sizeof(int)));
  checkCudaErrors(hipMalloc((void**)&dcooColIdx, nnz * sizeof(int)));
  checkCudaErrors(hipMalloc((void**)&dcooValSorted, nnz * sizeof(double)));
  checkCudaErrors(hipMalloc((void**)&buffer, bufferSize ));
  checkCudaErrors(hipMalloc((void**)&dP, nnz * sizeof(int)));

  checkCudaErrors(hipMemcpy(dcooVal, cooVal, nnz * sizeof(double), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(dcooRowIdx, cooRowIdx, nnz * sizeof(int), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(dcooColIdx, cooColIdx, nnz * sizeof(int), hipMemcpyHostToDevice));

  checkCudaErrors(hipsparseCreateIdentityPermutation(handle, nnz, dP));
  
  checkCudaErrors(hipsparseXcoosortByColumn(handle, m, n, nnz, 
                                        dcooRowIdx, dcooColIdx,
                                        dP, buffer));
  checkCudaErrors(hipsparseDgthr(handle, nnz, dcooVal, dcooValSorted, dP,
                                HIPSPARSE_INDEX_BASE_ZERO));

  checkCudaErrors(hipMemcpy(cooVal, dcooValSorted, nnz * sizeof(double), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(cooRowIdx, dcooRowIdx, nnz * sizeof(int), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(cooColIdx, dcooColIdx, nnz * sizeof(int), hipMemcpyDeviceToHost));

  checkCudaErrors(hipFree(dcooVal));
  checkCudaErrors(hipFree(dcooRowIdx));
  checkCudaErrors(hipFree(dcooColIdx));
  checkCudaErrors(hipFree(dcooValSorted));
  checkCudaErrors(hipFree(buffer));
  checkCudaErrors(hipFree(dP));

}

//coo sorted by row
void coo2csr_gpu(hipsparseHandle_t handle, hipStream_t stream, int m, int n, int nnz,
                double * cooVal, int * cooRowIdx, int * cooColIdx,
                double * csrVal, int * csrRowPtr, int * csrColIdx) {
  checkCudaErrors(hipsparseXcoo2csr(handle, cooRowIdx, nnz, m, csrRowPtr,
                  HIPSPARSE_INDEX_BASE_ZERO));
  checkCudaErrors(hipMemcpyAsync(csrVal, cooVal, nnz * sizeof(double), hipMemcpyDeviceToDevice, stream));
  checkCudaErrors(hipMemcpyAsync(csrColIdx, cooColIdx, nnz * sizeof(int), hipMemcpyDeviceToDevice, stream));
}

int findFirstInSorted(int * a, int n, int key) {
  int l = 0;
  int r = n;
  int m = l + (r - l) / 2;
  while (l < r - 1) {
    int m = l + (r - l) / 2;
    if (key <= a[m]) {
      r = m;
    } else if (key > a[m]) {
      l = m;
    } 
  }
  //cout << "a[" << l << "] = " <<  a[l] << endl;
  //cout << " a[" << r << "] = " << a[r] << endl;
  //cout << " idx = " << idx << endl;
  if (key == a[l]) return l;
  if (key == a[r]) return r;
  return l;
}


int findLastInSorted(int * a, int n, int key) {
  int l = 0;
  int r = n;
  int m = l + (r - l) / 2;
  while (l < r - 1) {
    int m = l + (r - l) / 2;
    if (key < a[m]) {
      r = m;
    } else if (key >= a[m]) {
      l = m;
    } 
  }
  //cout << "a[" << l << "] = " <<  a[l] << endl;
  //cout << " a[" << r << "] = " << a[r] << endl;
  //cout << " idx = " << idx << endl;
  if (key == a[l]) return l;
  if (key == a[r]) return r;
  return l;
}



