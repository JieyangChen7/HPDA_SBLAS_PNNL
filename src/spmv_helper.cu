#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <ctime>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "hipsparse.h"
#include <iostream>
#include <cmath>
#include <float.h>
#include "common_cuda.h"
//#include "anonymouslib_cuda.h"
#include "spmv_kernel.h"
#include <limits>
#include <sstream>
#include <string>
using namespace std;

int get_row_from_index(int n, int * a, int idx) {
  int l = 0;
  int r = n;
  int m = l + (r - l) / 2;
  while (l < r - 1) {
    //cout << "l = " << l <<endl;
    //cout << "r = " << r <<endl;
    int m = l + (r - l) / 2;
    //cout << "m = " << m <<endl;
    if (idx < a[m]) {
      r = m;
    } else if (idx >= a[m]) {
      l = m;
    } 
    //else {
    //  cout << "1st return: " << m << endl;
    //  return m;
    //}
  }
  //cout << "a[" << l << "] = " <<  a[l] << endl;
  //cout << " a[" << r << "] = " << a[r] << endl;
  //cout << " idx = " << idx << endl;
  if (idx == a[l]) return l;
  if (idx == a[r]) return r;
  return l;

}

double get_time()
{
  struct timeval tp;
  gettimeofday(&tp, NULL);
  double ms = (double)tp.tv_sec * 1000 + (double)tp.tv_usec / 1000; //get current timestamp in milliseconds
  //return 0.00001;
  return ms / 1000;
}


double get_gpu_availble_mem(int ngpu) {
  size_t uCurAvailMemoryInBytes;
  size_t uTotalMemoryInBytes;
  

  double min_mem = numeric_limits<double>::max();
  int device;
  for (device = 0; device < ngpu; ++device) 
  {
    hipSetDevice(device);
    hipMemGetInfo(&uCurAvailMemoryInBytes, &uTotalMemoryInBytes);
    cout << uCurAvailMemoryInBytes << "/" << uTotalMemoryInBytes << endl;
    double aval_mem = (double)uCurAvailMemoryInBytes/1e9;
    cout << aval_mem << endl;
    if (aval_mem < min_mem) {
      min_mem = aval_mem;
    }
      // hipDeviceProp_t deviceProp;
      // hipGetDeviceProperties(&deviceProp, device);
      // printf("Device %d has compute capability %d.%d.\n",
      //        device, deviceProp.major, deviceProp.minor);
  }


  return min_mem;
}

void print_vec(double * a, int n, string s) {
  ostringstream ss;
  ss << s << ": ";
  for (int i = 0; i < n ; i++) {
    ss << a[i] << " ";
  }
  ss << endl;
  string res = ss.str();
  cout << res;
}

void print_vec(int * a, int n, string s) {
  ostringstream ss;
  ss << s << ": ";
  for (int i = 0; i < n ; i++) {
    ss << a[i] << " ";
  }
  ss << endl;
  string res = ss.str();
  cout << res;
}



void print_vec_gpu(double * a, int n, string s) {
  double * ha = new double[n]; 
  hipMemcpy(ha, a, n*sizeof(double), hipMemcpyDeviceToHost);
  print_vec(ha, n, s);
  delete [] ha;
}

void print_vec_gpu(int * a, int n, string s) {
  int * ha = new int[n];
  hipMemcpy(ha, a, n*sizeof(int), hipMemcpyDeviceToHost);
  print_vec(ha, n, s);
  delete [] ha;
}


void coo2csr(int m, int n, int nnz,
             double * cooVal, int * cooRowIdx, int * cooColIdx,
             double * csrVal, int * csrRowPtr, int * csrColIdx) {
  double * dcooVal;
  int * dcooRowIdx;
  int * dcooColIdx;
  double * dcsrVal;
  int * dcsrRowPtr;
  int * dcsrColIdx;

  checkCudaErrors(hipMalloc((void**)&dcooVal, nnz * sizeof(double)));
  checkCudaErrors(hipMalloc((void**)&dcooRowIdx, nnz * sizeof(int)));
  checkCudaErrors(hipMalloc((void**)&dcooColIdx, nnz * sizeof(int)));

  checkCudaErrors(hipMalloc((void**)&dcsrVal, nnz * sizeof(double)));
  checkCudaErrors(hipMalloc((void**)&dcsrRowPtr, (m+1) * sizeof(int)));
  checkCudaErrors(hipMalloc((void**)&dcsrColIdx, nnz * sizeof(int)));

  checkCudaErrors(hipMemcpy(dcooVal, cooVal, nnz * sizeof(double),
                              hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(dcooRowIdx, cooRowIdx, nnz * sizeof(int),
                              hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(dcooColIdx, cooColIdx, nnz * sizeof(int),
                              hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(dcsrVal, csrVal, nnz * sizeof(double),
                              hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(dcsrRowPtr, csrRowPtr, (m+1) * sizeof(int),
                              hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(dcsrColIdx, csrColIdx, nnz * sizeof(int),
                              hipMemcpyHostToDevice));

  coo2csr_gpu(m, n, nnz,
              dcooVal, dcooRowIdx, dcooColIdx,
              dcsrVal, dcsrRowPtr, dcsrColIdx);

  checkCudaErrors(hipMemcpy(cooVal, dcooVal, nnz * sizeof(double),
                              hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(cooRowIdx, dcooRowIdx, nnz * sizeof(int),
                              hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(cooColIdx, dcooColIdx, nnz * sizeof(int),
                              hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(csrVal, dcsrVal, nnz * sizeof(double),
                              hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(csrRowPtr, dcsrRowPtr, (m+1) * sizeof(int),
                              hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(csrColIdx, dcsrColIdx, nnz * sizeof(int),
                              hipMemcpyDeviceToHost));

  checkCudaErrors(hipFree(dcooVal));
  checkCudaErrors(hipFree(dcooRowIdx));
  checkCudaErrors(hipFree(dcooColIdx));
  checkCudaErrors(hipFree(dcsrVal));
  checkCudaErrors(hipFree(dcsrRowPtr));
  checkCudaErrors(hipFree(dcsrColIdx));

}

void coo2csc(int m, int n, int nnz,
             double * cooVal, int * cooRowIdx, int * cooColIdx,
             double * cscVal, int * cscColPtr, int * cscRowIdx) {
  hipStream_t stream;
  hipsparseStatus_t status;
  hipsparseHandle_t handle;
  hipsparseMatDescr_t descr;
  hipStreamCreate(&stream);
  status = hipsparseCreate(&handle); 
  if (status != HIPSPARSE_STATUS_SUCCESS) 
  { 
    printf("CUSPARSE Library initialization failed");
    return; 
  } 
  status = hipsparseSetStream(handle, stream);
  if (status != HIPSPARSE_STATUS_SUCCESS) 
  { 
    printf("Stream bindind failed");
    return;
  } 
  status = hipsparseCreateMatDescr(&descr);
  if (status != HIPSPARSE_STATUS_SUCCESS) 
  { 
    printf("Matrix descriptor initialization failed");
    return;
  }   
  hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL); 
  hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO);


  hipsparseDestroyMatDescr(descr);
  hipsparseDestroy(handle);
  hipStreamDestroy(stream);

  
}

void coo2csr_gpu(int m, int n, int nnz,
                 double * cooVal, int * cooRowIdx, int * cooColIdx,
                 double * csrVal, int * csrRowPtr, int * csrColIdx) {
  hipStream_t stream;
  hipsparseStatus_t status;
  hipsparseHandle_t handle;
  hipsparseMatDescr_t descr;
  checkCudaErrors(hipStreamCreate(&stream));
  checkCudaErrors(hipsparseCreate(&handle)); 
  checkCudaErrors(hipsparseSetStream(handle, stream));
  checkCudaErrors(hipsparseCreateMatDescr(&descr));
  checkCudaErrors(hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL)); 
  checkCudaErrors(hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO));


  size_t buffer_size;
  checkCudaErrors(hipsparseXcoosort_bufferSizeExt(handle, m, n, nnz
                                                 cooRowIdx, cooColIdx,
                                                 &buffer_size));

  void * buffer;
  checkCudaErrors(hipMalloc((void**)&buffer, buffer_size));

  int * P = new int[nnz];
  checkCudaErrors(hipsparseXcoosortByRow(handle, m, n, nnz,
                                        cooRowIdx, cooColIdx,
                                        P, buffer));
  checkCudaErrors(hipsparseXcoo2csr(handle,
                                   cooRowIdx, nnz, m, 
                                   HIPSPARSE_INDEX_BASE_ZERO));
  checkCudaErrors(hipMemcpy(csrVal, cooVal, nnz * sizeof(double), 
                              hipMemcpyDeviceToDevice));
  checkCudaErrors(hipMemcpy(csrColIdx, cooColIdx, nnz * sizeof(double), 
                              hipMemcpyDeviceToDevice));

  checkCudaErrors(hipDeviceSynchronize());
  checkCudaErrors(hipFree(buffer));
  delete [] P;

  checkCudaErrors(hipsparseDestroyMatDescr(descr));
  checkCudaErrors(hipsparseDestroy(handle));
  checkCudaErrors(hipStreamDestroy(stream));

  
}

void csc2csr_gpu(int m, int n, int nnz,
                 double * cscVal, int * cscColPtr, int * cscRowIdx,
                 double * csrVal, int * csrRowPtr, int * csrColIdx) {
  hipStream_t stream;
  hipsparseStatus_t status;
  hipsparseHandle_t handle;
  hipsparseMatDescr_t descr;
  hipStreamCreate(&stream);
  status = hipsparseCreate(&handle); 
  if (status != HIPSPARSE_STATUS_SUCCESS) 
  { 
    printf("CUSPARSE Library initialization failed");
    return; 
  } 
  status = hipsparseSetStream(handle, stream);
  if (status != HIPSPARSE_STATUS_SUCCESS) 
  { 
    printf("Stream bindind failed");
    return;
  } 
  status = hipsparseCreateMatDescr(&descr);
  if (status != HIPSPARSE_STATUS_SUCCESS) 
  { 
    printf("Matrix descriptor initialization failed");
    return;
  }   
  hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL); 
  hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO);


  hipsparseDestroyMatDescr(descr);
  hipsparseDestroy(handle);
  hipStreamDestroy(stream);

  
}




