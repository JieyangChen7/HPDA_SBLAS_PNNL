#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <ctime>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "hipsparse.h"
#include <iostream>
#include <cmath>
#include <float.h>
#include "common_cuda.h"
//#include "anonymouslib_cuda.h"
#include "spmv_kernel.h"
#include <limits>
#include <sstream>
#include <string>
using namespace std;

int get_row_from_index(int n, int * a, int idx) {
  int l = 0;
  int r = n;
  int m = l + (r - l) / 2;
  while (l < r - 1) {
    //cout << "l = " << l <<endl;
    //cout << "r = " << r <<endl;
    int m = l + (r - l) / 2;
    //cout << "m = " << m <<endl;
    if (idx < a[m]) {
      r = m;
    } else if (idx >= a[m]) {
      l = m;
    } 
    //else {
    //  cout << "1st return: " << m << endl;
    //  return m;
    //}
  }
  //cout << "a[" << l << "] = " <<  a[l] << endl;
  //cout << " a[" << r << "] = " << a[r] << endl;
  //cout << " idx = " << idx << endl;
  if (idx == a[l]) return l;
  if (idx == a[r]) return r;
  return l;

}

double get_time()
{
  struct timeval tp;
  gettimeofday(&tp, NULL);
  double ms = (double)tp.tv_sec * 1000 + (double)tp.tv_usec / 1000; //get current timestamp in milliseconds
  //return 0.00001;
  return ms / 1000;
}


double get_gpu_availble_mem(int ngpu) {
  size_t uCurAvailMemoryInBytes;
  size_t uTotalMemoryInBytes;
  

  double min_mem = numeric_limits<double>::max();
  int device;
  for (device = 0; device < ngpu; ++device) 
  {
    hipSetDevice(device);
    hipMemGetInfo(&uCurAvailMemoryInBytes, &uTotalMemoryInBytes);
    cout << uCurAvailMemoryInBytes << "/" << uTotalMemoryInBytes << endl;
    double aval_mem = (double)uCurAvailMemoryInBytes/1e9;
    cout << aval_mem << endl;
    if (aval_mem < min_mem) {
      min_mem = aval_mem;
    }
      // hipDeviceProp_t deviceProp;
      // hipGetDeviceProperties(&deviceProp, device);
      // printf("Device %d has compute capability %d.%d.\n",
      //        device, deviceProp.major, deviceProp.minor);
  }


  return min_mem;
}

void print_vec(double * a, int n, string s) {
  ostringstream ss;
  ss << s << ": ";
  for (int i = 0; i < n ; i++) {
    ss << a[i] << " ";
  }
  ss << endl;
  string res = ss.str();
  cout << res;
}

void print_vec(int * a, int n, string s) {
  ostringstream ss;
  ss << s << ": ";
  for (int i = 0; i < n ; i++) {
    ss << a[i] << " ";
  }
  ss << endl;
  string res = ss.str();
  cout << res;
}



void print_vec_gpu(double * a, int n, string s) {
  double * ha = new double[n]; 
  hipMemcpy(ha, a, n*sizeof(double), hipMemcpyDeviceToHost);
  print_vec(ha, n, s);
  delete [] ha;
}

void print_vec_gpu(int * a, int n, string s) {
  int * ha = new int[n];
  hipMemcpy(ha, a, n*sizeof(int), hipMemcpyDeviceToHost);
  print_vec(ha, n, s);
  delete [] ha;
}


void csr2csrNcsc(int m, int n, int nnz,
             double * cooVal, int * cooRowIdx, int * cooColIdx,
             double * csrVal, int * csrRowPtr, int * csrColIdx,
             double * cscVal, int * cscColPtr, int * cscRowIdx) {

  printf("converting: %d, %d\n", m, n);

  sortCOORow(m, n, nnz, cooVal, cooRowIdx, cooColIdx);


  // double * A = new double[m * n];
  // for (int i = 0; i < m; i++) {
  //   for (int j = 0; j < n; j++) {
  //     A[i * n + j] = 0.0;
  //   }
  // }

  // for (int i = 0; i < nnz; i++) {
  //   A[cooRowIdx[i] * n + cooColIdx[i]] = cooVal[i];
  // }

  // for (int i = 0; i < m; i++) {
  //   print_vec(&(A[i * n]), n, "A");
  // }

  // int p = 0;
  // csrRowPtr[0] = 0;
  // for (int i = 0; i < m; i++) {
  //   for (int j = 0; j < n; j++) {
  //     if (A[i * n + j] != 0) {
  //       csrVal[p] = A[i * n + j];
  //       csrColIdx[p] = j;
  //       //printf("add %f, %d\n", csrVal[p], csrColIdx[p]);
  //       p++;
  //     }
  //   }
  //   //printf("add to row %d\n", i+1);
  //   csrRowPtr[i + 1] = p;
  //   //printf("row %d\n", p);
  // }

  

  // p = 0;
  // cscColPtr[0] = 0;
  // for (int j = 0; j < n; j++) {
  //   for (int i = 0; i < m; i++) {
  //     if (A[i * n + j] != 0) {
  //       cscVal[p] = A[i * n + j];
  //       cscRowIdx[p] = i;
  //       p++;
  //     }
  //   }
  //   cscColPtr[j + 1] = p;
  // }

  // printf("done converting\n");
  //delete [] A;

  // double * dcsrVal;
  // int * dcsrRowPtr;
  // int * dcsrColIdx;
  // double * dcscVal;
  // int * dcscColPtr;
  // int * dcscRowIdx;


  // checkCudaErrors(hipMalloc((void**)&dcsrVal, nnz * sizeof(double)));
  // checkCudaErrors(hipMalloc((void**)&dcsrRowPtr, (m+1) * sizeof(int)));
  // checkCudaErrors(hipMalloc((void**)&dcsrColIdx, nnz * sizeof(int)));

  // checkCudaErrors(hipMalloc((void**)&dcscVal, nnz * sizeof(double)));
  // checkCudaErrors(hipMalloc((void**)&dcscColPtr, (n+1) * sizeof(int)));
  // checkCudaErrors(hipMalloc((void**)&dcscRowIdx, nnz * sizeof(int)));

  // checkCudaErrors(hipMemcpy(dcsrVal, csrVal, nnz * sizeof(double),
  //                             hipMemcpyHostToDevice));
  // checkCudaErrors(hipMemcpy(dcsrRowPtr, csrRowPtr, (m+1) * sizeof(int),
  //                             hipMemcpyHostToDevice));
  // checkCudaErrors(hipMemcpy(dcsrColIdx, csrColIdx, nnz * sizeof(int),
  //                             hipMemcpyHostToDevice));
  // checkCudaErrors(hipMemcpy(dcscVal, cscVal, nnz * sizeof(double),
  //                             hipMemcpyHostToDevice));
  // checkCudaErrors(hipMemcpy(dcscColPtr, cscColPtr, (n+1) * sizeof(int),
  //                             hipMemcpyHostToDevice));
  // checkCudaErrors(hipMemcpy(dcscRowIdx, cscRowIdx, nnz * sizeof(int),
  //                             hipMemcpyHostToDevice));

  // csr2csc_gpu(m, n, nnz,
  //             dcsrVal, dcsrRowPtr, dcsrColIdx,
  //             dcscVal, dcscColPtr, dcscRowIdx);

  
  // checkCudaErrors(hipMemcpy(csrVal, dcsrVal, nnz * sizeof(double),
  //                             hipMemcpyDeviceToHost));
  // checkCudaErrors(hipMemcpy(csrRowPtr, dcsrRowPtr, (m+1) * sizeof(int),
  //                             hipMemcpyDeviceToHost));
  // checkCudaErrors(hipMemcpy(csrColIdx, dcsrColIdx, nnz * sizeof(int),
  //                             hipMemcpyDeviceToHost));
  // checkCudaErrors(hipMemcpy(dcscVal, cscVal, nnz * sizeof(double),
  //                             hipMemcpyDeviceToHost));
  // checkCudaErrors(hipMemcpy(dcscColPtr, cscColPtr, (n+1) * sizeof(int),
  //                             hipMemcpyDeviceToHost));
  // checkCudaErrors(hipMemcpy(dcscRowIdx, cscRowIdx, nnz * sizeof(int),
  //                             hipMemcpyDeviceToHost));

  // checkCudaErrors(hipFree(dcsrVal));
  // checkCudaErrors(hipFree(dcsrRowPtr));
  // checkCudaErrors(hipFree(dcsrColIdx));
  // checkCudaErrors(hipFree(dcscVal));
  // checkCudaErrors(hipFree(dcscColPtr));
  // checkCudaErrors(hipFree(dcscRowIdx));

}


void csc2csrGPU(hipsparseHandle_t handle, int m, int n, int nnz, double * A, int lda, 
                 double * cscVal, int * cscColPtr, int * cscRowIdx,
                 double * csrVal, int * csrRowPtr, int * csrColIdx) {
  
  hipsparseMatDescr_t descr;
  checkCudaErrors(hipsparseCreateMatDescr(&descr));
  checkCudaErrors(hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL)); 
  checkCudaErrors(hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO));

  int * nnzPerRow = new int[m];
  int * nnzTotalDevHostPtr = new int[1];
  checkCudaErrors(hipsparseDcsc2dense(handle, m, n, descr,
                                     cscVal, cscRowIdx, cscColPtr,
                                     A, lda));

  checkCudaErrors(hipsparseDnnz(handle, HIPSPARSE_DIRECTION_ROW,
                               m, n, descr, A, lda, nnzPerRow, nnzTotalDevHostPtr));

  checkCudaErrors(hipsparseDdense2csr(handle, m, n, descr, 
                                     A, lda, nnzPerRow,
                                     csrVal, csrRowPtr, csrColIdx));

  // checkCudaErrors(hipsparseDcsr2csc(handle, m, n, nnz,
  //                                   csrVal, csrRowPtr, csrColIdx,
  //                                   cscVal, cscColPtr, cscRowIdx,
  //                                   HIPSPARSE_ACTION_NUMERIC,
  //                                   HIPSPARSE_INDEX_BASE_ZERO));

  // checkCudaErrors(hipDeviceSynchronize());
  // checkCudaErrors(hipFree(buffer));
  // delete [] P;

  // checkCudaErrors(hipsparseDestroyMatDescr(descr));
  // checkCudaErrors(hipsparseDestroy(handle));
  // checkCudaErrors(hipStreamDestroy(stream));

  
}

void sortCOORow(int m, int n, int nnz,
                double * cooVal, int * cooRowIdx, int * cooColIdx) {
  hipStream_t stream;
  hipsparseStatus_t status;
  hipsparseHandle_t handle;

  checkCudaErrors(hipStreamCreate(&stream));
  checkCudaErrors(hipsparseCreate(&handle)); 
  checkCudaErrors(hipsparseSetStream(handle, stream));

  double * dcooVal;
  int * dcooRowIdx;
  int * dcooColIdx;
  double * dcooValSorted;
  void * buffer;
  size_t bufferSize = 0;
  int * dP;

  checkCudaErrors(hipsparseXcoosort_bufferSizeExt(handle, m, n, nnz, 
                                                  dcooRowIdx, dcooColIdx,
                                                  &bufferSize));

  checkCudaErrors(hipMalloc((void**)&dcooVal, nnz * sizeof(double)));
  checkCudaErrors(hipMalloc((void**)&dcooRowIdx, nnz * sizeof(int)));
  checkCudaErrors(hipMalloc((void**)&dcooColIdx, nnz * sizeof(int)));
  checkCudaErrors(hipMalloc((void**)&dcooValSorted, nnz * sizeof(double)));
  checkCudaErrors(hipMalloc((void**)&buffer, bufferSize ));
  checkCudaErrors(hipMalloc((void**)&dP, nnz * sizeof(int)));

  checkCudaErrors(hipMemcpy(dcooVal, cooVal, nnz * sizeof(double), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(dcooRowIdx, cooRowIdx, nnz * sizeof(int), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(dcooColIdx, cooColIdx, nnz * sizeof(int), hipMemcpyHostToDevice));

  checkCudaErrors(hipsparseCreateIdentityPermutation(handle, nnz, dP));
  
  checkCudaErrors(hipsparseXcoosortByRow(handle, m, n, nnz, 
                                        dcooRowIdx, dcooColIdx,
                                        dP, buffer));
  checkCudaErrors(hipsparseDgthr(handle, nnz, dcooVal, dcooValSorted, dP,
                                HIPSPARSE_INDEX_BASE_ZERO));
  checkCudaErrors(hipDeviceSynchronize());


  checkCudaErrors(hipMemcpy(cooVal, dcooValSorted, nnz * sizeof(double), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(cooRowIdx, dcooRowIdx, nnz * sizeof(int), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(cooColIdx, dcooColIdx, nnz * sizeof(int), hipMemcpyDeviceToHost));

  checkCudaErrors(hipFree(dcooVal));
  checkCudaErrors(hipFree(dcooRowIdx));
  checkCudaErrors(hipFree(dcooColIdx));
  checkCudaErrors(hipFree(dcooValSorted));
  checkCudaErrors(hipFree(buffer));
  checkCudaErrors(hipFree(dP));

}

void sortCOOCol(int m, int n, int nnz,
                double * cooVal, int * cooRowIdx, int * cooColIdx) {
  hipStream_t stream;
  hipsparseStatus_t status;
  hipsparseHandle_t handle;

  checkCudaErrors(hipStreamCreate(&stream));
  checkCudaErrors(hipsparseCreate(&handle)); 
  checkCudaErrors(hipsparseSetStream(handle, stream));

  double * dcooVal;
  int * dcooRowIdx;
  int * dcooColIdx;
  double * dcooValSorted;
  void * buffer;
  size_t bufferSize = 0;
  int * dP;

  checkCudaErrors(hipsparseXcoosort_bufferSizeExt(handle, m, n, nnz, 
                                                  dcooRowIdx, dcooColIdx,
                                                  &bufferSize));

  checkCudaErrors(hipMalloc((void**)&dcooVal, nnz * sizeof(double)));
  checkCudaErrors(hipMalloc((void**)&dcooRowIdx, nnz * sizeof(int)));
  checkCudaErrors(hipMalloc((void**)&dcooColIdx, nnz * sizeof(int)));
  checkCudaErrors(hipMalloc((void**)&dcooValSorted, nnz * sizeof(double)));
  checkCudaErrors(hipMalloc((void**)&buffer, bufferSize ));
  checkCudaErrors(hipMalloc((void**)&dP, nnz * sizeof(int)));

  checkCudaErrors(hipMemcpy(dcooVal, cooVal, nnz * sizeof(double), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(dcooRowIdx, cooRowIdx, nnz * sizeof(int), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(dcooColIdx, cooColIdx, nnz * sizeof(int), hipMemcpyHostToDevice));

  checkCudaErrors(hipsparseCreateIdentityPermutation(handle, nnz, dP));
  
  checkCudaErrors(hipsparseXcoosortByColumn(handle, m, n, nnz, 
                                        dcooRowIdx, dcooColIdx,
                                        dP, buffer));
  checkCudaErrors(hipsparseDgthr(handle, nnz, dcooVal, dcooValSorted, dP,
                                HIPSPARSE_INDEX_BASE_ZERO));
  checkCudaErrors(hipDeviceSynchronize());


  checkCudaErrors(hipMemcpy(cooVal, dcooValSorted, nnz * sizeof(double), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(cooRowIdx, dcooRowIdx, nnz * sizeof(int), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(cooColIdx, dcooColIdx, nnz * sizeof(int), hipMemcpyDeviceToHost));

  checkCudaErrors(hipFree(dcooVal));
  checkCudaErrors(hipFree(dcooRowIdx));
  checkCudaErrors(hipFree(dcooColIdx));
  checkCudaErrors(hipFree(dcooValSorted));
  checkCudaErrors(hipFree(buffer));
  checkCudaErrors(hipFree(dP));

}

void sortCOOGPUEx (hipsparseHandle_t handle, int m, int n, int nnz,
                      int * cooRowIdx, int * cooColIdx,
                      double ** cooValSorted,  void ** buffer, int ** P) {
  size_t bufferSize = 0;
  checkCudaErrors(hipsparseXcoosort_bufferSizeExt(handle, m, n, nnz, 
                                                 cooRowIdx, cooColIdx,
                                                 &bufferSize));


  checkCudaErrors(hipMalloc((void**)cooValSorted, nnz * sizeof(double)));
  checkCudaErrors(hipMalloc((void**)buffer, bufferSize ));
  checkCudaErrors(hipMalloc((void**)P, nnz * sizeof(int)));
}




void sortCOORowGPU(hipsparseHandle_t handle, hipStream_t stream,
                    int m, int n, int nnz,
                    double * cooVal, int * cooRowIdx, int * cooColIdx,
                    double * cooValSorted,  void * buffer, int * P) {
  

  // double * cooValSorted;
  // void * buffer;
  // size_t bufferSize = 0;
  // int * P;

  // checkCudaErrors(hipsparseXcoosort_bufferSizeExt(handle, m, n, nnz, 
  //                                                 dcooRowIdx, dcooColIdx,
  //                                                 &bufferSize));

  // checkCudaErrors(hipMalloc((void**)&cooValSorted, nnz * sizeof(double)));
  // checkCudaErrors(hipMalloc((void**)&buffer, bufferSize ));
  // checkCudaErrors(hipMalloc((void**)&P, nnz * sizeof(int)));

  checkCudaErrors(hipsparseCreateIdentityPermutation(handle, nnz, P));
  
  checkCudaErrors(hipsparseXcoosortByRow(handle, m, n, nnz, 
                                        cooRowIdx, cooColIdx,
                                        P, buffer));
  checkCudaErrors(hipsparseDgthr(handle, nnz, cooVal, cooValSorted, P,
                                HIPSPARSE_INDEX_BASE_ZERO));

  checkCudaErrors(hipMemcpyAsync(cooVal, cooValSorted, nnz * sizeof(double), 
                                  hipMemcpyDeviceToDevice, stream));

  // checkCudaErrors(hipFree(dcooValSorted));
  // checkCudaErrors(hipFree(buffer));
  // checkCudaErrors(hipFree(dP));

}

void sortCOOColGPU(hipsparseHandle_t handle, hipStream_t stream,
                int m, int n, int nnz,
                double * cooVal, int * cooRowIdx, int * cooColIdx,
                double * cooValSorted, void * buffer, int * P) {
 
  // double * cooValSorted;
  // void * buffer;
  // size_t bufferSize = 0;
  // int * P;

  // checkCudaErrors(hipsparseXcoosort_bufferSizeExt(handle, m, n, nnz, 
  //                                                 cooRowIdx, cooColIdx,
  //                                                 &bufferSize));

  // checkCudaErrors(hipMalloc((void**)&dcooValSorted, nnz * sizeof(double)));
  // checkCudaErrors(hipMalloc((void**)&buffer, bufferSize ));
  // checkCudaErrors(hipMalloc((void**)&P, nnz * sizeof(int)));

  checkCudaErrors(hipsparseCreateIdentityPermutation(handle, nnz, P));
  
  checkCudaErrors(hipsparseXcoosortByColumn(handle, m, n, nnz, 
                                        cooRowIdx, cooColIdx,
                                        P, buffer));
  checkCudaErrors(hipsparseDgthr(handle, nnz, cooVal, cooValSorted, P,
                                HIPSPARSE_INDEX_BASE_ZERO));

  checkCudaErrors(hipMemcpyAsync(cooVal, cooValSorted, nnz * sizeof(double), 
                                  hipMemcpyDeviceToDevice, stream));

  // checkCudaErrors(hipFree(dcooValSorted));
  // checkCudaErrors(hipFree(buffer));
  // checkCudaErrors(hipFree(dP));

}

//coo sorted by row
void coo2csrGPU(hipsparseHandle_t handle, hipStream_t stream, int m, int n, int nnz,
                double * cooVal, int * cooRowIdx, int * cooColIdx,
                double * csrVal, int * csrRowPtr, int * csrColIdx) {

  // checkCudaErrors(hipDeviceSynchronize());
  // print_vec_gpu(cooRowIdx, nnz, "cooRowIdx");
  checkCudaErrors(hipsparseXcoo2csr(handle, cooRowIdx, nnz, m, csrRowPtr,
                  HIPSPARSE_INDEX_BASE_ZERO));
  // checkCudaErrors(hipDeviceSynchronize());
  // print_vec_gpu(csrRowPtr, m+1, "csrRowPtr");
  checkCudaErrors(hipMemcpyAsync(csrVal, cooVal, nnz * sizeof(double), hipMemcpyDeviceToDevice, stream));
  checkCudaErrors(hipMemcpyAsync(csrColIdx, cooColIdx, nnz * sizeof(int), hipMemcpyDeviceToDevice, stream));
}


void coo2csr(int m, int n, int nnz,
             double * cooVal, int * cooRowIdx, int * cooColIdx,
             double * csrVal, int * csrRowPtr, int * csrColIdx) {
  hipStream_t stream;
  hipsparseStatus_t status;
  hipsparseHandle_t handle;

  checkCudaErrors(hipStreamCreate(&stream));
  checkCudaErrors(hipsparseCreate(&handle)); 
  checkCudaErrors(hipsparseSetStream(handle, stream));

  double * dcooVal;
  int * dcooRowIdx;
  int * dcooColIdx;
  checkCudaErrors(hipMalloc((void**)&dcooVal, nnz * sizeof(double)));
  checkCudaErrors(hipMalloc((void**)&dcooRowIdx, nnz * sizeof(int)));
  checkCudaErrors(hipMalloc((void**)&dcooColIdx, nnz * sizeof(int)));

  double * dcsrVal;
  int * dcsrRowPtr;
  int * dcsrColIdx;
  checkCudaErrors(hipMalloc((void**)&dcsrVal, nnz * sizeof(double)));
  checkCudaErrors(hipMalloc((void**)&dcsrRowPtr, (m+1) * sizeof(int)));
  checkCudaErrors(hipMalloc((void**)&dcsrColIdx, nnz * sizeof(int)));


  checkCudaErrors(hipMemcpy(dcooVal, cooVal, nnz * sizeof(double), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(dcooRowIdx, cooRowIdx, nnz * sizeof(int), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(dcooColIdx, cooColIdx, nnz * sizeof(int), hipMemcpyHostToDevice));

  double * cooValSorted;
  void * buffer;
  int * P;

  sortCOOGPUEx(handle, m, n, nnz,
               dcooRowIdx, dcooColIdx,
               &cooValSorted, &buffer, &P);
  checkCudaErrors(hipDeviceSynchronize());

  sortCOORowGPU(handle, stream,
                 m, n, nnz,
                 dcooVal, dcooRowIdx, dcooColIdx,
                 cooValSorted, buffer, P);
  checkCudaErrors(hipDeviceSynchronize());

  // print_vec_gpu(dcooVal, 5, "dcooVal");
  // print_vec_gpu(dcooRowIdx, 5, "dcooRowIdx");
  // print_vec_gpu(dcooColIdx, 5, "dcooColIdx");

  coo2csrGPU(handle, stream, m, n, nnz,
             dcooVal, dcooRowIdx, dcooColIdx,
             dcsrVal, dcsrRowPtr, dcsrColIdx);

  checkCudaErrors(hipDeviceSynchronize());

  // print_vec_gpu(dcsrVal, 5, "dcsrVal");
  // print_vec_gpu(dcsrRowPtr+m, 1, "dcsrRowPtr");
  // print_vec_gpu(dcsrColIdx, 5, "dcsrColIdx");

  checkCudaErrors(hipMemcpy(csrVal, dcsrVal,       nnz * sizeof(double), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(csrRowPtr, dcsrRowPtr, (m+1) * sizeof(int), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(csrColIdx, dcsrColIdx, nnz * sizeof(int), hipMemcpyDeviceToHost));

  checkCudaErrors(hipDeviceSynchronize());

  // print_vec(csrRowPtr+m, 1, "csrRowPtr");

  checkCudaErrors(hipFree(dcooVal));
  checkCudaErrors(hipFree(dcooRowIdx));
  checkCudaErrors(hipFree(dcooColIdx));

  checkCudaErrors(hipFree(dcsrVal));
  checkCudaErrors(hipFree(dcsrRowPtr));
  checkCudaErrors(hipFree(dcsrColIdx));

  checkCudaErrors(hipFree(cooValSorted));
  checkCudaErrors(hipFree(buffer));
  checkCudaErrors(hipFree(P));

}


void coo2csc(int m, int n, int nnz,
             double * cooVal, int * cooRowIdx, int * cooColIdx,
             double * cscVal, int * cscColPtr, int * cscRowIdx) {
  hipStream_t stream;
  hipsparseStatus_t status;
  hipsparseHandle_t handle;

  checkCudaErrors(hipStreamCreate(&stream));
  checkCudaErrors(hipsparseCreate(&handle)); 
  checkCudaErrors(hipsparseSetStream(handle, stream));

  double * dcooVal;
  int * dcooRowIdx;
  int * dcooColIdx;
  checkCudaErrors(hipMalloc((void**)&dcooVal, nnz * sizeof(double)));
  checkCudaErrors(hipMalloc((void**)&dcooRowIdx, nnz * sizeof(int)));
  checkCudaErrors(hipMalloc((void**)&dcooColIdx, nnz * sizeof(int)));

  double * dcscVal;
  int * dcscColPtr;
  int * dcscRowIdx;
  checkCudaErrors(hipMalloc((void**)&dcscVal, nnz * sizeof(double)));
  checkCudaErrors(hipMalloc((void**)&dcscColPtr, (n+1) * sizeof(int)));
  checkCudaErrors(hipMalloc((void**)&dcscRowIdx, nnz * sizeof(int)));

  checkCudaErrors(hipMemcpy(dcooVal, cooVal, nnz * sizeof(double), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(dcooRowIdx, cooRowIdx, nnz * sizeof(int), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(dcooColIdx, cooColIdx, nnz * sizeof(int), hipMemcpyHostToDevice));

  double * cooValSorted;
  void * buffer;
  int * P;

  sortCOOGPUEx(handle, m, n, nnz,
               dcooRowIdx, dcooColIdx,
               &cooValSorted, &buffer, &P);
  checkCudaErrors(hipDeviceSynchronize());

  sortCOOColGPU(handle, stream,
                 m, n, nnz,
                 dcooVal, dcooRowIdx, dcooColIdx,
                 cooValSorted, buffer, P);
  checkCudaErrors(hipDeviceSynchronize());

  coo2csrGPU(handle, stream, n, m, nnz,
             dcooVal, dcooColIdx, dcooRowIdx,
             dcscVal, dcscColPtr, dcscRowIdx);
  checkCudaErrors(hipDeviceSynchronize());

  checkCudaErrors(hipMemcpy(cscVal, dcscVal,       nnz * sizeof(double), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(cscColPtr, dcscColPtr, (n+1) * sizeof(int), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(cscRowIdx, dcscRowIdx, nnz * sizeof(int), hipMemcpyDeviceToHost));

  checkCudaErrors(hipDeviceSynchronize());

  checkCudaErrors(hipFree(dcooVal));
  checkCudaErrors(hipFree(dcooRowIdx));
  checkCudaErrors(hipFree(dcooColIdx));

  checkCudaErrors(hipFree(dcscVal));
  checkCudaErrors(hipFree(dcscColPtr));
  checkCudaErrors(hipFree(dcscRowIdx));

  checkCudaErrors(hipFree(cooValSorted));
  checkCudaErrors(hipFree(buffer));
  checkCudaErrors(hipFree(P));

}



int findFirstInSorted(int * a, int n, int key) {
  int l = 0;
  int r = n;
  int m = l + (r - l) / 2;
  while (l < r - 1) {
    int m = l + (r - l) / 2;
    if (key <= a[m]) {
      r = m;
    } else if (key > a[m]) {
      l = m;
    } 
  }
  //cout << "a[" << l << "] = " <<  a[l] << endl;
  //cout << " a[" << r << "] = " << a[r] << endl;
  //cout << " idx = " << idx << endl;
  if (key == a[l]) return l;
  if (key == a[r]) return r;
  return l;
}


int findLastInSorted(int * a, int n, int key) {
  int l = 0;
  int r = n;
  int m = l + (r - l) / 2;
  while (l < r - 1) {
    int m = l + (r - l) / 2;
    if (key < a[m]) {
      r = m;
    } else if (key >= a[m]) {
      l = m;
    } 
  }
  //cout << "a[" << l << "] = " <<  a[l] << endl;
  //cout << " a[" << r << "] = " << a[r] << endl;
  //cout << " idx = " << idx << endl;
  if (key == a[l]) return l;
  if (key == a[r]) return r;
  return l;
}

void report_mem_usage(int d) {
  size_t free_byte ;
  size_t total_byte ;
  checkCudaErrors(hipSetDevice(d));
  checkCudaErrors(hipMemGetInfo( &free_byte, &total_byte ));
  double free_db = (double)free_byte ;
  double total_db = (double)total_byte ;
  double used_db = total_db - free_db ;
  printf("GPU %d memory usage: used = %f, free = %f MB, total = %f MB\n", d,
  used_db/1024.0/1024.0, free_db/1024.0/1024.0, total_db/1024.0/1024.0);
}
void report_all_mem_usage() {
  int deviceCount;
  checkCudaErrors(hipGetDeviceCount(&deviceCount));
  for (int d = 0; d < deviceCount; d++) {
    report_mem_usage(d);
  }

