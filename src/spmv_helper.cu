#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <ctime>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "hipsparse.h"
#include <iostream>
#include <cmath>
#include <float.h>
#include "common_cuda.h"
//#include "anonymouslib_cuda.h"
#include "spmv_kernel.h"
#include <limits>
#include <sstream>
#include <string>
using namespace std;

int get_row_from_index(int n, int * a, int idx) {
  int l = 0;
  int r = n;
  int m = l + (r - l) / 2;
  while (l < r - 1) {
    //cout << "l = " << l <<endl;
    //cout << "r = " << r <<endl;
    int m = l + (r - l) / 2;
    //cout << "m = " << m <<endl;
    if (idx < a[m]) {
      r = m;
    } else if (idx >= a[m]) {
      l = m;
    } 
    //else {
    //  cout << "1st return: " << m << endl;
    //  return m;
    //}
  }
  //cout << "a[" << l << "] = " <<  a[l] << endl;
  //cout << " a[" << r << "] = " << a[r] << endl;
  //cout << " idx = " << idx << endl;
  if (idx == a[l]) return l;
  if (idx == a[r]) return r;
  return l;

}

double get_time()
{
  struct timeval tp;
  gettimeofday(&tp, NULL);
  double ms = (double)tp.tv_sec * 1000 + (double)tp.tv_usec / 1000; //get current timestamp in milliseconds
  //return 0.00001;
  return ms / 1000;
}


double get_gpu_availble_mem(int ngpu) {
  size_t uCurAvailMemoryInBytes;
  size_t uTotalMemoryInBytes;
  

  double min_mem = numeric_limits<double>::max();
  int device;
  for (device = 0; device < ngpu; ++device) 
  {
    hipSetDevice(device);
    hipMemGetInfo(&uCurAvailMemoryInBytes, &uTotalMemoryInBytes);
    cout << uCurAvailMemoryInBytes << "/" << uTotalMemoryInBytes << endl;
    double aval_mem = (double)uCurAvailMemoryInBytes/1e9;
    cout << aval_mem << endl;
    if (aval_mem < min_mem) {
      min_mem = aval_mem;
    }
      // hipDeviceProp_t deviceProp;
      // hipGetDeviceProperties(&deviceProp, device);
      // printf("Device %d has compute capability %d.%d.\n",
      //        device, deviceProp.major, deviceProp.minor);
  }


  return min_mem;
}

void print_vec(double * a, int n, string s) {
  ostringstream ss;
  ss << s << ": ";
  for (int i = 0; i < n ; i++) {
    ss << a[i] << " ";
  }
  ss << endl;
  string res = ss.str();
  cout << res;
}

void print_vec(int * a, int n, string s) {
  ostringstream ss;
  ss << s << ": ";
  for (int i = 0; i < n ; i++) {
    ss << a[i] << " ";
  }
  ss << endl;
  string res = ss.str();
  cout << res;
}



void print_vec_gpu(double * a, int n, string s) {
  double * ha = new double[n]; 
  hipMemcpy(ha, a, n*sizeof(double), hipMemcpyDeviceToHost);
  print_vec(ha, n, s);
  delete [] ha;
}

void print_vec_gpu(int * a, int n, string s) {
  int * ha = new int[n];
  hipMemcpy(ha, a, n*sizeof(int), hipMemcpyDeviceToHost);
  print_vec(ha, n, s);
  delete [] ha;
}


void csr2csrNcsc(int m, int n, int nnz,
             double * cooVal, int * cooRowIdx, int * cooColIdx,
             double * csrVal, int * csrRowPtr, int * csrColIdx,
             double * cscVal, int * cscColPtr, int * cscRowIdx) {

  double * A = new double[m * n];
  for (int i = 0; i < m; i++) {
    for (int j = 0; j < n; j++) {
      A[i * m + j] = 0.0;
    }
  }

  for (int i = 0; i < nnz; i++) {
    A[cooRowIdx[i] * m + cooColIdx[i]] = cooVal[i];
  }

  int p = 0;
  csrRowPtr[0] = 0;
  for (int i = 0; i < m; i++) {
    for (int j = 0; j < n; j++) {
      if (A[i * m + j] != 0) {
        csrVal[p] = A[i * m + j];
        csrColIdx[p] = j;
        printf("add %f, %d\n", csrVal[p], csrColIdx[p]);
        p++;
      }
    }
    printf("add to row %d\n", i+1);
    csrRowPtr[i + 1] = p;
    printf("row %d\n", p);
  }

  printf("")

  // p = 0;
  // cscColPtr[0] = 0;
  // for (int j = 0; j < n; j++) {
  //   for (int i = 0; i < m; i++) {
  //     if (A[i * m + j] != 0) {
  //       cscVal[p] = A[i * m + j];
  //       cscRowIdx[p] = i;
  //       p++;
  //     }
  //   }
  //   cscColPtr[j + 1] = p;
  // }



  // double * dcsrVal;
  // int * dcsrRowPtr;
  // int * dcsrColIdx;
  // double * dcscVal;
  // int * dcscColPtr;
  // int * dcscRowIdx;


  // checkCudaErrors(hipMalloc((void**)&dcsrVal, nnz * sizeof(double)));
  // checkCudaErrors(hipMalloc((void**)&dcsrRowPtr, (m+1) * sizeof(int)));
  // checkCudaErrors(hipMalloc((void**)&dcsrColIdx, nnz * sizeof(int)));

  // checkCudaErrors(hipMalloc((void**)&dcscVal, nnz * sizeof(double)));
  // checkCudaErrors(hipMalloc((void**)&dcscColPtr, (n+1) * sizeof(int)));
  // checkCudaErrors(hipMalloc((void**)&dcscRowIdx, nnz * sizeof(int)));

  // checkCudaErrors(hipMemcpy(dcsrVal, csrVal, nnz * sizeof(double),
  //                             hipMemcpyHostToDevice));
  // checkCudaErrors(hipMemcpy(dcsrRowPtr, csrRowPtr, (m+1) * sizeof(int),
  //                             hipMemcpyHostToDevice));
  // checkCudaErrors(hipMemcpy(dcsrColIdx, csrColIdx, nnz * sizeof(int),
  //                             hipMemcpyHostToDevice));
  // checkCudaErrors(hipMemcpy(dcscVal, cscVal, nnz * sizeof(double),
  //                             hipMemcpyHostToDevice));
  // checkCudaErrors(hipMemcpy(dcscColPtr, cscColPtr, (n+1) * sizeof(int),
  //                             hipMemcpyHostToDevice));
  // checkCudaErrors(hipMemcpy(dcscRowIdx, cscRowIdx, nnz * sizeof(int),
  //                             hipMemcpyHostToDevice));

  // csr2csc_gpu(m, n, nnz,
  //             dcsrVal, dcsrRowPtr, dcsrColIdx,
  //             dcscVal, dcscColPtr, dcscRowIdx);

  
  // checkCudaErrors(hipMemcpy(csrVal, dcsrVal, nnz * sizeof(double),
  //                             hipMemcpyDeviceToHost));
  // checkCudaErrors(hipMemcpy(csrRowPtr, dcsrRowPtr, (m+1) * sizeof(int),
  //                             hipMemcpyDeviceToHost));
  // checkCudaErrors(hipMemcpy(csrColIdx, dcsrColIdx, nnz * sizeof(int),
  //                             hipMemcpyDeviceToHost));
  // checkCudaErrors(hipMemcpy(dcscVal, cscVal, nnz * sizeof(double),
  //                             hipMemcpyDeviceToHost));
  // checkCudaErrors(hipMemcpy(dcscColPtr, cscColPtr, (n+1) * sizeof(int),
  //                             hipMemcpyDeviceToHost));
  // checkCudaErrors(hipMemcpy(dcscRowIdx, cscRowIdx, nnz * sizeof(int),
  //                             hipMemcpyDeviceToHost));

  // checkCudaErrors(hipFree(dcsrVal));
  // checkCudaErrors(hipFree(dcsrRowPtr));
  // checkCudaErrors(hipFree(dcsrColIdx));
  // checkCudaErrors(hipFree(dcscVal));
  // checkCudaErrors(hipFree(dcscColPtr));
  // checkCudaErrors(hipFree(dcscRowIdx));

}


void csr2csc(int m, int n, int nnz,
             double * csrVal, int * csrRowPtr, int * csrColIdx,
             double * cscVal, int * cscColPtr, int * cscRowIdx) {

  



  double * dcsrVal;
  int * dcsrRowPtr;
  int * dcsrColIdx;
  double * dcscVal;
  int * dcscColPtr;
  int * dcscRowIdx;


  checkCudaErrors(hipMalloc((void**)&dcsrVal, nnz * sizeof(double)));
  checkCudaErrors(hipMalloc((void**)&dcsrRowPtr, (m+1) * sizeof(int)));
  checkCudaErrors(hipMalloc((void**)&dcsrColIdx, nnz * sizeof(int)));

  checkCudaErrors(hipMalloc((void**)&dcscVal, nnz * sizeof(double)));
  checkCudaErrors(hipMalloc((void**)&dcscColPtr, (n+1) * sizeof(int)));
  checkCudaErrors(hipMalloc((void**)&dcscRowIdx, nnz * sizeof(int)));

  checkCudaErrors(hipMemcpy(dcsrVal, csrVal, nnz * sizeof(double),
                              hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(dcsrRowPtr, csrRowPtr, (m+1) * sizeof(int),
                              hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(dcsrColIdx, csrColIdx, nnz * sizeof(int),
                              hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(dcscVal, cscVal, nnz * sizeof(double),
                              hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(dcscColPtr, cscColPtr, (n+1) * sizeof(int),
                              hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(dcscRowIdx, cscRowIdx, nnz * sizeof(int),
                              hipMemcpyHostToDevice));

  csr2csc_gpu(m, n, nnz,
              dcsrVal, dcsrRowPtr, dcsrColIdx,
              dcscVal, dcscColPtr, dcscRowIdx);

  
  checkCudaErrors(hipMemcpy(csrVal, dcsrVal, nnz * sizeof(double),
                              hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(csrRowPtr, dcsrRowPtr, (m+1) * sizeof(int),
                              hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(csrColIdx, dcsrColIdx, nnz * sizeof(int),
                              hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(dcscVal, cscVal, nnz * sizeof(double),
                              hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(dcscColPtr, cscColPtr, (n+1) * sizeof(int),
                              hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(dcscRowIdx, cscRowIdx, nnz * sizeof(int),
                              hipMemcpyDeviceToHost));

  checkCudaErrors(hipFree(dcsrVal));
  checkCudaErrors(hipFree(dcsrRowPtr));
  checkCudaErrors(hipFree(dcsrColIdx));
  checkCudaErrors(hipFree(dcscVal));
  checkCudaErrors(hipFree(dcscColPtr));
  checkCudaErrors(hipFree(dcscRowIdx));

}



void csr2csc_gpu(int m, int n, int nnz,
                 double * csrVal, int * csrRowPtr, int * csrColIdx,
                 double * cscVal, int * cscColPtr, int * cscRowIdx) {
  hipStream_t stream;
  hipsparseStatus_t status;
  hipsparseHandle_t handle;
  hipsparseMatDescr_t descr;
  checkCudaErrors(hipStreamCreate(&stream));
  checkCudaErrors(hipsparseCreate(&handle)); 
  checkCudaErrors(hipsparseSetStream(handle, stream));
  checkCudaErrors(hipsparseCreateMatDescr(&descr));
  checkCudaErrors(hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL)); 
  checkCudaErrors(hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO));

  double * A;
  int lda = m;
  checkCudaErrors(hipMalloc((void**)&A, lda * n * sizeof(double)));

  int * nnzPerCol = new int[n];
  int * nnzTotalDevHostPtr = new int[1];
  checkCudaErrors(hipsparseDcsr2dense(handle, m, n, descr,
                                     csrVal, csrRowPtr, csrColIdx,
                                     A, lda));

  checkCudaErrors(hipsparseDnnz(handle, HIPSPARSE_DIRECTION_COLUMN,
                               m, n, descr, A, lda, nnzPerCol, nnzTotalDevHostPtr));

  checkCudaErrors(hipsparseDdense2csc(handle, m, n, descr, 
                                     A, lda, nnzPerCol,
                                     cscVal, cscColPtr, cscRowIdx));

  // checkCudaErrors(hipsparseDcsr2csc(handle, m, n, nnz,
  //                                   csrVal, csrRowPtr, csrColIdx,
  //                                   cscVal, cscColPtr, cscRowIdx,
  //                                   HIPSPARSE_ACTION_NUMERIC,
  //                                   HIPSPARSE_INDEX_BASE_ZERO));

  checkCudaErrors(hipDeviceSynchronize());
  // checkCudaErrors(hipFree(buffer));
  // delete [] P;

  checkCudaErrors(hipsparseDestroyMatDescr(descr));
  checkCudaErrors(hipsparseDestroy(handle));
  checkCudaErrors(hipStreamDestroy(stream));

  
}




